#include "hip/hip_runtime.h"
// **************************************************************************
//                                dipole_lj.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the dipole/cut pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( q_tex,float);
_texture( mu_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture( q_tex,int2);
_texture_2d( mu_tex,int4);
#endif

#else
#define pos_tex x_
#define q_tex q_
#define mu_tex mu_
#endif

#if (SHUFFLE_AVAIL == 0)

#define store_answers_tq(f, tor, energy, e_coul, virial, ii, inum, tid,     \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add6(t_per_atom, red_acc, offset, tid, f.x, f.y, f.z,       \
                     tor.x, tor.y, tor.z);                                  \
    if (EVFLAG && (vflag==2 || eflag==2)) {                                 \
      if (eflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_add2(t_per_atom, red_acc, offset, tid, energy, e_coul); \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_arr(6, t_per_atom, red_acc, offset, tid, virial);       \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }                                                                         \
  if (EVFLAG && (eflag || vflag)) {                                         \
    int ei=BLOCK_ID_X;                                                      \
    if (eflag!=2 && vflag!=2) {                                             \
      const int ev_stride=NUM_BLOCKS_X;                                     \
      if (eflag) {                                                          \
        simdsync();                                                         \
        block_reduce_add2(simd_size(), red_acc, tid, energy, e_coul);       \
        if (vflag) __syncthreads();                                         \
        if (tid==0) {                                                       \
          engv[ei]=energy*(acctyp)0.5;                                      \
          ei+=ev_stride;                                                    \
          engv[ei]=e_coul*(acctyp)0.5;                                      \
          ei+=ev_stride;                                                    \
        }                                                                   \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        block_reduce_arr(6, simd_size(), red_acc, tid, virial);             \
        if (tid==0) {                                                       \
          for (int r=0; r<6; r++) {                                         \
            engv[ei]=virial[r]*(acctyp)0.5;                                 \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (EVFLAG && eflag) {                                                \
        engv[ei]=energy*(acctyp)0.5;                                        \
        ei+=inum;                                                           \
        engv[ei]=e_coul*(acctyp)0.5;                                        \
        ei+=inum;                                                           \
      }                                                                     \
      if (EVFLAG && vflag) {                                                \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]=virial[i]*(acctyp)0.5;                                   \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

#else

#if (EVFLAG == 1)

#define store_answers_tq(f, tor, energy, e_coul, virial, ii, inum, tid,     \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add6(t_per_atom, f.x, f.y, f.z, tor.x, tor.y, tor.z);       \
    if (vflag==2 || eflag==2) {                                             \
      if (eflag)                                                            \
        simd_reduce_add2(t_per_atom,energy,e_coul);                         \
      if (vflag)                                                            \
        simd_reduce_arr(6, t_per_atom,virial);                              \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }                                                                         \
  if (eflag || vflag) {                                                     \
    if (eflag!=2 && vflag!=2) {                                             \
      const int vwidth = simd_size();                                       \
      const int voffset = tid & (simd_size() - 1);                          \
      const int bnum = tid/simd_size();                                     \
      int active_subgs = BLOCK_SIZE_X/simd_size();                          \
      for ( ; active_subgs > 1; active_subgs /= vwidth) {                   \
        if (active_subgs < BLOCK_SIZE_X/simd_size()) __syncthreads();       \
        if (bnum < active_subgs) {                                          \
          if (eflag) {                                                      \
            simd_reduce_add2(vwidth, energy, e_coul);                       \
            if (voffset==0) {                                               \
              red_acc[6][bnum] = energy;                                    \
              red_acc[7][bnum] = e_coul;                                    \
            }                                                               \
          }                                                                 \
          if (vflag) {                                                      \
            simd_reduce_arr(6, vwidth, virial);                             \
            if (voffset==0)                                                 \
              for (int r=0; r<6; r++) red_acc[r][bnum]=virial[r];           \
          }                                                                 \
        }                                                                   \
                                                                            \
        __syncthreads();                                                    \
        if (tid < active_subgs) {                                           \
          if (eflag) {                                                      \
            energy = red_acc[6][tid];                                       \
            e_coul = red_acc[7][tid];                                       \
          }                                                                 \
          if (vflag)                                                        \
            for (int r = 0; r < 6; r++) virial[r] = red_acc[r][tid];        \
        } else {                                                            \
          if (eflag) energy = e_coul = (acctyp)0;                           \
          if (vflag) for (int r = 0; r < 6; r++) virial[r] = (acctyp)0;     \
        }                                                                   \
      }                                                                     \
                                                                            \
      if (bnum == 0) {                                                      \
        int ei=BLOCK_ID_X;                                                  \
        const int ev_stride=NUM_BLOCKS_X;                                   \
        if (eflag) {                                                        \
          simd_reduce_add2(vwidth, energy, e_coul);                         \
          if (tid==0) {                                                     \
            engv[ei]=energy*(acctyp)0.5;                                    \
            ei+=ev_stride;                                                  \
            engv[ei]=e_coul*(acctyp)0.5;                                    \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
        if (vflag) {                                                        \
          simd_reduce_arr(6, vwidth, virial);                               \
          if (tid==0) {                                                     \
            for (int r=0; r<6; r++) {                                       \
              engv[ei]=virial[r]*(acctyp)0.5;                               \
              ei+=ev_stride;                                                \
            }                                                               \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (eflag) {                                                          \
        engv[ei]=energy*(acctyp)0.5;                                        \
        ei+=inum;                                                           \
        engv[ei]=e_coul*(acctyp)0.5;                                        \
        ei+=inum;                                                           \
      }                                                                     \
      if (vflag) {                                                          \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]=virial[i]*(acctyp)0.5;                                   \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

#else

#define store_answers_tq(f, tor, energy, e_coul, virial, ii, inum, tid,     \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1)                                                         \
    simd_reduce_add6(t_per_atom, f.x, f.y, f.z, tor.x, tor.y, tor.z);       \
  if (offset==0 && ii<inum) {                                               \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }

#endif
#endif

__kernel void k_dipole_lj(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict lj1,
                          const __global numtyp4 *restrict lj3,
                          const int lj_types,
                          const __global numtyp *restrict sp_lj_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const __global numtyp4 *restrict mu_,
                          const __global numtyp *restrict cutsq,
                          const numtyp qqrd2e, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  int n_stride;
  local_allocate_store_charge();

  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp4 f, tor;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  tor.x=(acctyp)0; tor.y=(acctyp)0; tor.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv, r7inv;
        numtyp pre1, pre2, pre3, pre4;
        numtyp pdotp, pidotr, pjdotr;
        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*r3inv;

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
                  r7inv = r5inv*r2inv;
            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            pre1 = (numtyp)3.0*r5inv*pdotp - (numtyp)15.0*r7inv*pidotr*pjdotr;
            pre2 = (numtyp)3.0*r5inv*pjdotr;
            pre3 = (numtyp)3.0*r5inv*pidotr;
            pre4 = (numtyp)(-1.0)*r3inv;

            forcecoul.x += pre1*delx + pre2*mui.x + pre3*muj.x;
            forcecoul.y += pre1*dely + pre2*mui.y + pre3*muj.y;
            forcecoul.z += pre1*delz + pre2*mui.z + pre3*muj.z;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pre1 = (numtyp)3.0*qj*r5inv * pidotr;
            pre2 = qj*r3inv;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;
            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr;
            pre2 = qtmp*r3inv;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;
        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (EVFLAG && eflag) {
          acctyp e = (acctyp)0.0;
          if (rsq < lj1[mtype].w) {
            e = qtmp*qj*rinv;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr;
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr;
            e *= fq;
          } else e = (acctyp)0.0;
            e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          }
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }

    } // for nbor
  } // if ii
  store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,
                   eflag,vflag,ans,engv);
}

__kernel void k_dipole_lj_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp4 *restrict lj1_in,
                               const __global numtyp4 *restrict lj3_in,
                               const __global numtyp *restrict sp_lj_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const __global numtyp4 *restrict mu_,
                               const __global numtyp *restrict _cutsq,
                               const numtyp qqrd2e, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  int n_stride;
  local_allocate_store_charge();

  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    cutsq[tid]=_cutsq[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }

  acctyp4 f, tor;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  tor.x=(acctyp)0; tor.y=(acctyp)0; tor.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv, r7inv;
        numtyp pre1, pre2, pre3, pre4;
        numtyp pdotp, pidotr, pjdotr;
        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*r3inv;

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            r7inv = r5inv*r2inv;
            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            pre1 = (numtyp)3.0*r5inv*pdotp - (numtyp)15.0*r7inv*pidotr*pjdotr;
            pre2 = (numtyp)3.0*r5inv*pjdotr;
            pre3 = (numtyp)3.0*r5inv*pidotr;
            pre4 = (numtyp)(-1.0)*r3inv;

            forcecoul.x += pre1*delx + pre2*mui.x + pre3*muj.x;
            forcecoul.y += pre1*dely + pre2*mui.y + pre3*muj.y;
            forcecoul.z += pre1*delz + pre2*mui.z + pre3*muj.z;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pre1 = (numtyp)3.0*qj*r5inv * pidotr;
            pre2 = qj*r3inv;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;
            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr;
            pre2 = qtmp*r3inv;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;

        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (EVFLAG && eflag) {
          acctyp e = (acctyp)0;
          if (rsq < lj1[mtype].w) {
            e = qtmp*qj*rinv;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr;
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr;
            e *= fq;
          } else e = (acctyp)0;
          e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          }
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }

    } // for nbor
  } // if ii
  store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,
                   eflag,vflag,ans,engv);
}
