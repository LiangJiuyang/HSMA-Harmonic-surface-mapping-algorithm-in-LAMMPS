#include "hip/hip_runtime.h"
// **************************************************************************
//                                   gauss.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the gauss pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif
#else
#define pos_tex x_
#endif

__kernel void k_gauss(const __global numtyp4 *restrict x_,
                      const __global numtyp4 *restrict gauss1,
                      const int lj_types,
                      const __global int *dev_nbor,
                      const __global int *dev_packed,
                      __global acctyp4 *restrict ans,
                      __global acctyp *restrict engv,
                      const int eflag, const int vflag, const int inum,
                      const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<gauss1[mtype].z) {
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_sqrt(rsq);
        numtyp force = (numtyp)-2.0*gauss1[mtype].x*gauss1[mtype].y*rsq*
        ucl_exp(-gauss1[mtype].y*rsq)*r2inv; //*factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e=-(gauss1[mtype].x*ucl_exp(-gauss1[mtype].y*rsq) -
            gauss1[mtype].w);
          energy+=e; //factor_lj*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_gauss_fast(const __global numtyp4 *restrict x_,
                           const __global numtyp4 *restrict gauss1_in,
                           const __global int *dev_nbor,
                           const __global int *dev_packed,
                           __global acctyp4 *restrict ans,
                           __global acctyp *restrict engv,
                           const int eflag, const int vflag, const int inum,
                           const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 gauss1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  int n_stride;
  local_allocate_store_pair();

  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    gauss1[tid]=gauss1_in[tid];
  }

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<gauss1[mtype].z) {
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_sqrt(rsq);
        numtyp force = (numtyp)-2.0*gauss1[mtype].x*gauss1[mtype].y*rsq*
        ucl_exp(-gauss1[mtype].y*rsq)*r2inv; //*factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e=-(gauss1[mtype].x*ucl_exp(-gauss1[mtype].y*rsq) -
            gauss1[mtype].w);
          energy+=e; //factor_lj*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

