#include "hip/hip_runtime.h"
// **************************************************************************
//                               lj_tip4p_long.cu
//                             -------------------
//                              V. Nikolskiy (HSE)
//
//  Device code for acceleration of the lj/tip4p/long pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : thevsevak@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)

#include "lal_aux_fun1.h"
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#include "inttypes.h"
#define tagint int64_t
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( q_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( q_tex,int2);
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

ucl_inline int atom_mapping(const __global int *map, tagint glob) {
  return map[glob];
}

ucl_inline int closest_image(int i, int j, const __global int* sametag,
                             const __global numtyp4 *restrict x_)
{
  if (j < 0) return j;

  numtyp4 xi; fetch4(xi,i,pos_tex); // = x[i];
  numtyp4 xj; fetch4(xj,j,pos_tex);

  int closest = j;
  numtyp delx = xi.x - xj.x;
  numtyp dely = xi.y - xj.y;
  numtyp delz = xi.z - xj.z;
  numtyp rsqmin = delx*delx + dely*dely + delz*delz;
  numtyp rsq;

  while (sametag[j] >= 0) {
    j = sametag[j];
    fetch4(xj,j,pos_tex);
    delx = xi.x - xj.x;
    dely = xi.y - xj.y;
    delz = xi.z - xj.z;
    rsq = delx*delx + dely*dely + delz*delz;
    if (rsq < rsqmin) {
      rsqmin = rsq;
      closest = j;
    }
  }

  return closest;
}

ucl_inline void compute_newsite(int iO, int  iH1, int  iH2,
    __global numtyp4 *xM, numtyp q,
    numtyp alpha, const __global numtyp4 *restrict x_) {
  numtyp4 xO;  fetch4(xO,iO,pos_tex);
  numtyp4 xH1; fetch4(xH1,iH1,pos_tex);
  numtyp4 xH2; fetch4(xH2,iH2,pos_tex);
  numtyp4 M;

  numtyp delx1 = xH1.x - xO.x;
  numtyp dely1 = xH1.y - xO.y;
  numtyp delz1 = xH1.z - xO.z;

  numtyp delx2 = xH2.x - xO.x;
  numtyp dely2 = xH2.y - xO.y;
  numtyp delz2 = xH2.z - xO.z;

  numtyp ap = alpha * (numtyp)0.5;

  M.x = xO.x + ap * (delx1 + delx2);
  M.y = xO.y + ap * (dely1 + dely2);
  M.z = xO.z + ap * (delz1 + delz2);
  M.w = q;

  *xM = M;
}

__kernel void k_lj_tip4p_long_distrib(const __global numtyp4 *restrict x_,
    __global acctyp4 *restrict ans,
    __global acctyp *restrict engv,
    const int eflag, const int vflag, const int inum,
    const int nbor_pitch, const int t_per_atom,
    __global int *restrict hneigh,
    __global numtyp4 *restrict m,
    const int typeO, const int typeH,
    const numtyp alpha,
    const __global numtyp *restrict q_, const __global acctyp4 *restrict ansO) {

  int i = BLOCK_ID_X*(BLOCK_SIZE_X)+THREAD_ID_X;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;

  if (i<inum) {
    numtyp4 ix; fetch4(ix,i,pos_tex);// = x_[i];
    int itype = ix.w;
    acctyp4 fM, vM;
    acctyp eM;
    // placement of the virial in engv depends on eflag value
    int engv_iter = eflag ? 2 : 0;
    if (itype == typeH) {
      int iO = hneigh[i*4];
      if (iO < inum) {
        fM = ansO[iO];
        f.x += fM.x * (acctyp)0.5 * alpha;
        f.y += fM.y * (acctyp)0.5 * alpha;
        f.z += fM.z * (acctyp)0.5 * alpha;
        if (EVFLAG && vflag) {
          vM = ansO[inum  +iO];
          engv[inum*engv_iter + i] += vM.x * (acctyp)0.5 * alpha; engv_iter++;
          engv[inum*engv_iter + i] += vM.y * (acctyp)0.5 * alpha; engv_iter++;
          engv[inum*engv_iter + i] += vM.z * (acctyp)0.5 * alpha; engv_iter++;
          vM = ansO[inum*2+iO];
          engv[inum*engv_iter + i] += vM.x * (acctyp)0.5 * alpha; engv_iter++;
          engv[inum*engv_iter + i] += vM.y * (acctyp)0.5 * alpha; engv_iter++;
          engv[inum*engv_iter + i] += vM.z * (acctyp)0.5 * alpha;
        }
      }
    } else {
      fM = ansO[i];
      int iH1 = hneigh[i*4  ];
      int iH2 = hneigh[i*4+1];
      f.x += fM.x * (acctyp)(1 - alpha);
      f.y += fM.y * (acctyp)(1 - alpha);
      f.z += fM.z * (acctyp)(1 - alpha);
      if (EVFLAG && eflag) {
        eM = engv[i+inum];
        engv[inum+i] = eM*(acctyp)(1 - alpha);
        if (iH1 < inum) engv[inum+iH1] += eM * (acctyp)0.5 * alpha;
        if (iH2 < inum) engv[inum+iH2] += eM * (acctyp)0.5 * alpha;
      }
      if (EVFLAG && vflag) {
        vM = ansO[inum   + i];
        engv[inum*engv_iter + i] += vM.x * (acctyp)(1 - alpha); engv_iter++;
        engv[inum*engv_iter + i] += vM.y * (acctyp)(1 - alpha); engv_iter++;
        engv[inum*engv_iter + i] += vM.z * (acctyp)(1 - alpha); engv_iter++;
        vM = ansO[inum*2 + i];
        engv[inum*engv_iter + i] += vM.x * (acctyp)(1 - alpha); engv_iter++;
        engv[inum*engv_iter + i] += vM.y * (acctyp)(1 - alpha); engv_iter++;
        engv[inum*engv_iter + i] += vM.z * (acctyp)(1 - alpha);
      }
    }
    acctyp4 old=ans[i];
    old.x+=f.x;
    old.y+=f.y;
    old.z+=f.z;
    ans[i]=old;
  } // if ii
}

__kernel void k_lj_tip4p_reneigh(const __global numtyp4 *restrict x_,
    const __global int * dev_nbor,
    const __global int * dev_packed,
    const int nall, const int inum,
    const int nbor_pitch, const int t_per_atom,
    __global int *restrict hneigh,
    __global numtyp4 *restrict m,
    const int typeO, const int typeH,
    const __global tagint *restrict tag, const __global int *restrict map,
    const __global int *restrict sametag) {

  int i = BLOCK_ID_X*(BLOCK_SIZE_X)+THREAD_ID_X;

  if (i<nall) {
    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    int iH1, iH2, iO;
    int itype = ix.w;
    if(itype == typeO) {
      iO  = i;
      if (hneigh[i*4+2] != -1) {
        iH1 = atom_mapping(map, tag[i] + 1);
        iH2 = atom_mapping(map, tag[i] + 2);
        // set iH1,iH2 to closest image to O
        iH1 = closest_image(i, iH1, sametag, x_);
        iH2 = closest_image(i, iH2, sametag, x_);
        hneigh[i*4  ] = iH1;
        hneigh[i*4+1] = iH2;
        hneigh[i*4+2] = -1;
      }
    } else {
      if (hneigh[i*4+2] != -1) {
        int iI, iH;
        iI = atom_mapping(map,tag[i] - 1);
        numtyp4 iIx; fetch4(iIx,iI,pos_tex); //x_[iI];
        if ((int)iIx.w == typeH) {
          iO = atom_mapping(map,tag[i] - 2);
          iO  = closest_image(i, iO, sametag, x_);
          iH1 = closest_image(i, iI, sametag, x_);
          iH2 = i;
        } else { //if ((int)iIx.w == typeO)
          iH = atom_mapping(map, tag[i] + 1);
          iO  = closest_image(i,iI,sametag, x_);
          iH1 = i;
          iH2 = closest_image(i,iH,sametag, x_);
        }
        hneigh[i*4+0] = iO;
        hneigh[i*4+1] += -1;
        hneigh[i*4+2] = -1;
      }
    }
  }
}


__kernel void k_lj_tip4p_newsite(const __global numtyp4 *restrict x_,
    const __global int * dev_nbor,
    const __global int * dev_packed,
    const int nall, const int inum,
    const int nbor_pitch, const int t_per_atom,
    __global int *restrict hneigh,
    __global numtyp4 *restrict m,
    const int typeO, const int typeH,
    const numtyp alpha, const __global numtyp *restrict q_) {

  int i = BLOCK_ID_X*(BLOCK_SIZE_X)+THREAD_ID_X;

  if (i<nall) {
    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype = ix.w;
    if (itype == typeO) {
      int iH1, iH2, iO;
      iH1 = hneigh[i*4  ];
      iH2 = hneigh[i*4+1];
      iO  = i;
      numtyp qO; fetch(qO,iO,q_tex);
      compute_newsite(iO,iH1,iH2, &m[iO], qO, alpha, x_);
    }
  }
}

__kernel void k_lj_tip4p_long(const __global numtyp4 *restrict x_,
    const __global numtyp4 *restrict lj1,
    const __global numtyp4 *restrict lj3,
    const int lj_types,
    const __global numtyp *restrict sp_lj,
    const __global int * dev_nbor,
    const __global int * dev_packed,
    __global acctyp4 *restrict ans,
    __global acctyp *restrict engv,
    const int eflag, const int vflag, const int inum,
    const int nbor_pitch, const int t_per_atom,
    __global int *restrict hneigh,
    __global numtyp4 *restrict m,
    const int typeO, const int typeH,
    const numtyp alpha,
    const __global numtyp *restrict q_,
    const __global numtyp *restrict cutsq,
    const numtyp qqrd2e, const numtyp g_ewald,
    const numtyp cut_coulsq, const numtyp cut_coulsqplus,
    __global acctyp4 *restrict ansO) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_charge();

  acctyp4 f, fO;
  f.x=(acctyp)0;  f.y=(acctyp)0;  f.z=(acctyp)0;
  fO.x=(acctyp)0; fO.y=(acctyp)0; fO.z=(acctyp)0;
  acctyp energy, e_coul, virial[6], vO[6];
  if (EVFLAG) {
    energy = (acctyp)0;
    e_coul = (acctyp)0;
    for (int i=0; i<6; i++) {
      virial[i]=(acctyp)0;
      vO[i]=(acctyp)0;
    }
  }

  int i;
  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype = ix.w;
    numtyp4 x1 = ix;

    int non_local_oxy = 0;
    int iH1, iH2, iO;

    if(itype == typeO) {
      iO  = i;
      iH1 = hneigh[i*4  ];
      iH2 = hneigh[i*4+1];
      x1 = m[iO];
    } else {
      iO  = hneigh[i *4  ];
      iH1 = hneigh[iO*4  ];
      iH2 = hneigh[iO*4+1];
      if (iO >= inum) {
        non_local_oxy = 1;
      }
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj,factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype = jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype = itype*lj_types+jtype;
      if (rsq < lj1[mtype].z) { // cut_ljsq
        numtyp r2inv = ucl_recip(rsq);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp forcelj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        forcelj *= r2inv*factor_lj;

        f.x += delx*forcelj;
        f.y += dely*forcelj;
        f.z += delz*forcelj;

        if (EVFLAG && eflag) {
          numtyp e = r6inv * (lj3[mtype].x*r6inv-lj3[mtype].y);
          energy += factor_lj * (e - lj3[mtype].z);
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*forcelj;
          virial[1] += dely*dely*forcelj;
          virial[2] += delz*delz*forcelj;
          virial[3] += delx*dely*forcelj;
          virial[4] += delx*delz*forcelj;
          virial[5] += dely*delz*forcelj;
        }
      } // if LJ

      if (rsq < cut_coulsqplus) { //cut_coulsqplus
        int jH1, jH2, jO;
        numtyp qj; fetch(qj,j,q_tex);
        numtyp4 x2 = jx;
        if(itype == typeO || jtype == typeO) {
          if (jtype == typeO) {
            jO = j;
            jH1 = hneigh[j*4  ];
            jH2 = hneigh[j*4+1];
            x2 = m[j];
          }
          delx = x1.x-x2.x;
          dely = x1.y-x2.y;
          delz = x1.z-x2.z;
          rsq = delx*delx+dely*dely+delz*delz;
        }
        if (rsq < cut_coulsq) {
          numtyp r2inv = ucl_recip(rsq);
          numtyp r = ucl_rsqrt(r2inv);
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          numtyp _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;

          numtyp prefactor = qj;
          prefactor *= qqrd2e*qtmp/r;
          numtyp force_coul = r2inv*prefactor * (_erfc + EWALD_F*grij*expm2 - factor_coul);

          if (itype == typeH) {
            f.x += delx * force_coul;
            f.y += dely * force_coul;
            f.z += delz * force_coul;
            f.w += 0;
          } else {
            fO.x += delx * force_coul;
            fO.y += dely * force_coul;
            fO.z += delz * force_coul;
            fO.w += 0;
          }
          if (EVFLAG && eflag) {
            e_coul += prefactor*(_erfc-factor_coul);
          }
          if (EVFLAG && vflag) {
            acctyp4 fd;
            fd.x = delx*force_coul;
            fd.y = dely*force_coul;
            fd.z = delz*force_coul;
            if (itype == typeH) {
              if (jtype == typeH) {
                virial[0] += delx*fd.x;
                virial[1] += dely*fd.y;
                virial[2] += delz*fd.z;
                virial[3] += delx*fd.y;
                virial[4] += delx*fd.z;
                virial[5] += dely*fd.z;
              } else {
                numtyp cO = 1 - alpha, cH = 0.5*alpha;
                numtyp4 vdj;
                numtyp4 xjH1; fetch4(xjH1,jH1,pos_tex);
                numtyp4 xjH2; fetch4(xjH2,jH2,pos_tex);
                numtyp4 xjO; fetch4(xjO,jO,pos_tex);
                vdj.x = xjO.x*cO + xjH1.x*cH + xjH2.x*cH;
                vdj.y = xjO.y*cO + xjH1.y*cH + xjH2.y*cH;
                vdj.z = xjO.z*cO + xjH1.z*cH + xjH2.z*cH;
                //vdj.w = vdj.w;
                virial[0] += (ix.x - vdj.x)*fd.x;
                virial[1] += (ix.y - vdj.y)*fd.y;
                virial[2] += (ix.z - vdj.z)*fd.z;
                virial[3] += (ix.x - vdj.x)*fd.y;
                virial[4] += (ix.x - vdj.x)*fd.z;
                virial[5] += (ix.y - vdj.y)*fd.z;
              }
            } else {
              numtyp cO = 1 - alpha, cH = 0.5*alpha;
              numtyp4 vdi, vdj;
              numtyp4 xH1; fetch4(xH1,iH1,pos_tex);
              numtyp4 xH2; fetch4(xH2,iH2,pos_tex);
              numtyp4 xO; fetch4(xO,iO,pos_tex);
              vdi.x = xO.x*cO + xH1.x*cH + xH2.x*cH;
              vdi.y = xO.y*cO + xH1.y*cH + xH2.y*cH;
              vdi.z = xO.z*cO + xH1.z*cH + xH2.z*cH;
              //vdi.w = vdi.w;
              if (jtype != typeH) {
                numtyp4 xjH1; fetch4(xjH1,jH1,pos_tex);
                numtyp4 xjH2; fetch4(xjH2,jH2,pos_tex);
                numtyp4 xjO; fetch4(xjO,jO,pos_tex);
                vdj.x = xjO.x*cO + xjH1.x*cH + xjH2.x*cH;
                vdj.y = xjO.y*cO + xjH1.y*cH + xjH2.y*cH;
                vdj.z = xjO.z*cO + xjH1.z*cH + xjH2.z*cH;
                //vdj.w = vdj.w;
              } else vdj = jx;
              vO[0] += 0.5*(vdi.x - vdj.x)*fd.x;
              vO[1] += 0.5*(vdi.y - vdj.y)*fd.y;
              vO[2] += 0.5*(vdi.z - vdj.z)*fd.z;
              vO[3] += 0.5*(vdi.x - vdj.x)*fd.y;
              vO[4] += 0.5*(vdi.x - vdj.x)*fd.z;
              vO[5] += 0.5*(vdi.y - vdj.y)*fd.z;
            }
          }
        }
        if (non_local_oxy == 1) {
          if (iO == j) {
            x2 = ix;
            qj = qtmp;
          }
          numtyp4 x1m = m[iO];
          delx = x1m.x-x2.x;
          dely = x1m.y-x2.y;
          delz = x1m.z-x2.z;
          rsq = delx*delx+dely*dely+delz*delz;
          if (rsq < cut_coulsq) {
            numtyp r2inv = ucl_recip(rsq);
            numtyp r = ucl_rsqrt(r2inv);
            numtyp grij = g_ewald * r;
            numtyp expm2 = ucl_exp(-grij*grij);
            numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
            numtyp _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;

            numtyp prefactor = qj;
            prefactor *= qqrd2e*x1m.w/r;
            numtyp force_coul = r2inv*prefactor * (_erfc + EWALD_F*grij*expm2 - factor_coul);

            numtyp cO = 1 - alpha, cH = 0.5*alpha;
            numtyp4 fd;
            fd.x = delx * force_coul * cH;
            fd.y = dely * force_coul * cH;
            fd.z = delz * force_coul * cH;

            f.x += fd.x;
            f.y += fd.y;
            f.z += fd.z;

            if (EVFLAG && eflag) {
              e_coul += prefactor*(_erfc-factor_coul) * (acctyp)0.5 * alpha;
            }
            if (EVFLAG && vflag) {
              numtyp4 xH1; fetch4(xH1,iH1,pos_tex);
              numtyp4 xH2; fetch4(xH2,iH2,pos_tex);
              numtyp4 xO;  fetch4(xO,iO,pos_tex);

              virial[0] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.x;
              virial[1] += ((xO.y*cO + xH1.y*cH + xH2.y*cH) - x2.y) * fd.y;
              virial[2] += ((xO.z*cO + xH1.z*cH + xH2.z*cH) - x2.z) * fd.z;
              virial[3] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.y;
              virial[4] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.z;
              virial[5] += ((xO.y*cO + xH1.y*cH + xH2.y*cH) - x2.y) * fd.z;
            }
          }
        }
      } // if cut_coulsqplus
    } // for nbor
  } // if ii
  if (t_per_atom>1) {
#if (SHUFFLE_AVAIL == 0)
    red_acc[0][tid]=fO.x;
    red_acc[1][tid]=fO.y;
    red_acc[2][tid]=fO.z;
    red_acc[3][tid]=fO.w;
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      simdsync();
      if (offset < s) {
        for (int r=0; r<4; r++)
          red_acc[r][tid] += red_acc[r][tid+s];
      }
    }
    fO.x=red_acc[0][tid];
    fO.y=red_acc[1][tid];
    fO.z=red_acc[2][tid];
    fO.w=red_acc[3][tid];
    if (EVFLAG && vflag) {
      simdsync();
      for (int r=0; r<6; r++) red_acc[r][tid]=vO[r];
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        simdsync();
        if (offset < s) {
          for (int r=0; r<6; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
      for (int r=0; r<6; r++) vO[r]=red_acc[r][tid];
    }
#else
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      fO.x += shfl_down(fO.x, s, t_per_atom);
      fO.y += shfl_down(fO.y, s, t_per_atom);
      fO.z += shfl_down(fO.z, s, t_per_atom);
      fO.w += shfl_down(fO.w, s, t_per_atom);
    }
    if (EVFLAG && vflag) {
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        for (int r=0; r<6; r++)
          vO[r] += shfl_down(vO[r], s, t_per_atom);
      }
    }
#endif
  }
  if(offset == 0 && ii<inum) {
    ansO[i] = fO;
    if (EVFLAG && vflag) {
      ansO[inum   + i].x = vO[0];
      ansO[inum   + i].y = vO[1];
      ansO[inum   + i].z = vO[2];
      ansO[inum*2 + i].x = vO[3];
      ansO[inum*2 + i].y = vO[4];
      ansO[inum*2 + i].z = vO[5];
    }
  }
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}

__kernel void k_lj_tip4p_long_fast(const __global numtyp4 *restrict x_,
    const __global numtyp4 *restrict lj1_in,
    const __global numtyp4 *restrict lj3_in,
    const int lj_types,
    const __global numtyp *restrict sp_lj_in,
    const __global int * dev_nbor,
    const __global int * dev_packed,
    __global acctyp4 *restrict ans,
    __global acctyp *restrict engv,
    const int eflag, const int vflag, const int inum,
    const int nbor_pitch, const int t_per_atom,
    __global int *restrict hneigh,
    __global numtyp4 *restrict m,
    const int typeO, const int typeH,
    const numtyp alpha,
    const __global numtyp *restrict q_,
    const __global numtyp *restrict cutsq,
    const numtyp qqrd2e, const numtyp g_ewald,
    const numtyp cut_coulsq, const numtyp cut_coulsqplus,
    __global acctyp4 *restrict ansO) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  int n_stride;
  local_allocate_store_charge();

  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }
  acctyp4 f, fO;
  f.x=(acctyp)0;  f.y=(acctyp)0;  f.z=(acctyp)0;
  fO.x=(acctyp)0; fO.y=(acctyp)0; fO.z=(acctyp)0;
  acctyp energy, e_coul, virial[6], vO[6];
  if (EVFLAG) {
    energy = (acctyp)0;
    e_coul = (acctyp)0;
    for (int i=0; i<6; i++) {
      virial[i]=(acctyp)0;
      vO[i]=(acctyp)0;
    }
  }

  __syncthreads();
  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
        n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype = ix.w;
    numtyp4 x1 = ix;

    int non_local_oxy = 0;
    int iH1, iH2, iO;

    if(itype == typeO) {
      iO  = i;
      iH1 = hneigh[i*4  ];
      iH2 = hneigh[i*4+1];
      x1 = m[iO];
    } else {
      iO  = hneigh[i *4  ];
      iH1 = hneigh[iO*4  ];
      iH2 = hneigh[iO*4+1];
      if (iO >= inum) {
        non_local_oxy = 1;
      }
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj,factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype = jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype = itype*lj_types+jtype;
      if (rsq < lj1[mtype].z) { // cut_ljsq
        numtyp r2inv = ucl_recip(rsq);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp forcelj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        forcelj *= r2inv*factor_lj;

        f.x += delx*forcelj;
        f.y += dely*forcelj;
        f.z += delz*forcelj;

        if (EVFLAG && eflag) {
          numtyp e = r6inv * (lj3[mtype].x*r6inv-lj3[mtype].y);
          energy += factor_lj * (e - lj3[mtype].z);
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*forcelj;
          virial[1] += dely*dely*forcelj;
          virial[2] += delz*delz*forcelj;
          virial[3] += delx*dely*forcelj;
          virial[4] += delx*delz*forcelj;
          virial[5] += dely*delz*forcelj;
        }
      } // if LJ

      if (rsq < cut_coulsqplus) { //cut_coulsqplus
        int jH1, jH2, jO;
        numtyp qj; fetch(qj,j,q_tex);
        numtyp4 x2 = jx;
        if(itype == typeO || jtype == typeO) {
          if (jtype == typeO) {
            jO = j;
            jH1 = hneigh[j*4  ];
            jH2 = hneigh[j*4+1];
            x2 = m[j];
          }
          delx = x1.x-x2.x;
          dely = x1.y-x2.y;
          delz = x1.z-x2.z;
          rsq = delx*delx+dely*dely+delz*delz;
        }
        if (rsq < cut_coulsq) {
          numtyp r2inv = ucl_recip(rsq);
          numtyp r = ucl_rsqrt(r2inv);
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          numtyp _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;

          numtyp prefactor = qj;
          prefactor *= qqrd2e*qtmp/r;
          numtyp force_coul = r2inv*prefactor * (_erfc + EWALD_F*grij*expm2 - factor_coul);

          if (itype == typeH) {
            f.x += delx * force_coul;
            f.y += dely * force_coul;
            f.z += delz * force_coul;
            f.w += 0;
          } else {
            fO.x += delx * force_coul;
            fO.y += dely * force_coul;
            fO.z += delz * force_coul;
            fO.w += 0;
          }
          if (EVFLAG && eflag) {
            e_coul += prefactor*(_erfc-factor_coul);
          }
          if (EVFLAG && vflag) {
            acctyp4 fd;
            fd.x = delx*force_coul;
            fd.y = dely*force_coul;
            fd.z = delz*force_coul;
            if (itype == typeH) {
              if (jtype == typeH) {
                virial[0] += delx*fd.x;
                virial[1] += dely*fd.y;
                virial[2] += delz*fd.z;
                virial[3] += delx*fd.y;
                virial[4] += delx*fd.z;
                virial[5] += dely*fd.z;
              } else {
                numtyp cO = 1 - alpha, cH = 0.5*alpha;
                numtyp4 vdj;
                numtyp4 xjH1; fetch4(xjH1,jH1,pos_tex);
                numtyp4 xjH2; fetch4(xjH2,jH2,pos_tex);
                numtyp4 xjO; fetch4(xjO,jO,pos_tex);
                vdj.x = xjO.x*cO + xjH1.x*cH + xjH2.x*cH;
                vdj.y = xjO.y*cO + xjH1.y*cH + xjH2.y*cH;
                vdj.z = xjO.z*cO + xjH1.z*cH + xjH2.z*cH;
                //vdj.w = vdj.w;
                virial[0] += (ix.x - vdj.x)*fd.x;
                virial[1] += (ix.y - vdj.y)*fd.y;
                virial[2] += (ix.z - vdj.z)*fd.z;
                virial[3] += (ix.x - vdj.x)*fd.y;
                virial[4] += (ix.x - vdj.x)*fd.z;
                virial[5] += (ix.y - vdj.y)*fd.z;
              }
            } else {
              numtyp cO = 1 - alpha, cH = 0.5*alpha;
              numtyp4 vdi, vdj;
              numtyp4 xH1; fetch4(xH1,iH1,pos_tex);
              numtyp4 xH2; fetch4(xH2,iH2,pos_tex);
              numtyp4 xO; fetch4(xO,iO,pos_tex);
              vdi.x = xO.x*cO + xH1.x*cH + xH2.x*cH;
              vdi.y = xO.y*cO + xH1.y*cH + xH2.y*cH;
              vdi.z = xO.z*cO + xH1.z*cH + xH2.z*cH;
              //vdi.w = vdi.w;
              if (jtype != typeH) {
                numtyp4 xjH1; fetch4(xjH1,jH1,pos_tex);
                numtyp4 xjH2; fetch4(xjH2,jH2,pos_tex);
                numtyp4 xjO; fetch4(xjO,jO,pos_tex);
                vdj.x = xjO.x*cO + xjH1.x*cH + xjH2.x*cH;
                vdj.y = xjO.y*cO + xjH1.y*cH + xjH2.y*cH;
                vdj.z = xjO.z*cO + xjH1.z*cH + xjH2.z*cH;
                //vdj.w = vdj.w;
              } else vdj = jx;
              vO[0] += 0.5*(vdi.x - vdj.x)*fd.x;
              vO[1] += 0.5*(vdi.y - vdj.y)*fd.y;
              vO[2] += 0.5*(vdi.z - vdj.z)*fd.z;
              vO[3] += 0.5*(vdi.x - vdj.x)*fd.y;
              vO[4] += 0.5*(vdi.x - vdj.x)*fd.z;
              vO[5] += 0.5*(vdi.y - vdj.y)*fd.z;
            }
          }
        }
        if (non_local_oxy == 1) {
          if (iO == j) {
            x2 = ix;
            qj = qtmp;
          }
          numtyp4 x1m = m[iO];
          delx = x1m.x-x2.x;
          dely = x1m.y-x2.y;
          delz = x1m.z-x2.z;
          rsq = delx*delx+dely*dely+delz*delz;
          if (rsq < cut_coulsq) {
            numtyp r2inv = ucl_recip(rsq);
            numtyp r = ucl_rsqrt(r2inv);
            numtyp grij = g_ewald * r;
            numtyp expm2 = ucl_exp(-grij*grij);
            numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
            numtyp _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;

            numtyp prefactor = qj;
            prefactor *= qqrd2e*x1m.w/r;
            numtyp force_coul = r2inv*prefactor * (_erfc + EWALD_F*grij*expm2 - factor_coul);

            numtyp cO = 1 - alpha, cH = 0.5*alpha;
            numtyp4 fd;
            fd.x = delx * force_coul * cH;
            fd.y = dely * force_coul * cH;
            fd.z = delz * force_coul * cH;

            f.x += fd.x;
            f.y += fd.y;
            f.z += fd.z;

            if (EVFLAG && eflag) {
              e_coul += prefactor*(_erfc-factor_coul) * (acctyp)0.5 * alpha;
            }
            if (EVFLAG && vflag) {
              numtyp4 xH1; fetch4(xH1,iH1,pos_tex);
              numtyp4 xH2; fetch4(xH2,iH2,pos_tex);
              numtyp4 xO;  fetch4(xO,iO,pos_tex);

              virial[0] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.x;
              virial[1] += ((xO.y*cO + xH1.y*cH + xH2.y*cH) - x2.y) * fd.y;
              virial[2] += ((xO.z*cO + xH1.z*cH + xH2.z*cH) - x2.z) * fd.z;
              virial[3] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.y;
              virial[4] += ((xO.x*cO + xH1.x*cH + xH2.x*cH) - x2.x) * fd.z;
              virial[5] += ((xO.y*cO + xH1.y*cH + xH2.y*cH) - x2.y) * fd.z;
            }
          }
        }
      } // if cut_coulsqplus
    } // for nbor
    if (t_per_atom>1) {
#if (SHUFFLE_AVAIL == 0)
      red_acc[0][tid]=fO.x;
      red_acc[1][tid]=fO.y;
      red_acc[2][tid]=fO.z;
      red_acc[3][tid]=fO.w;
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        simdsync();
        if (offset < s) {
          for (int r=0; r<4; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
      fO.x=red_acc[0][tid];
      fO.y=red_acc[1][tid];
      fO.z=red_acc[2][tid];
      fO.w=red_acc[3][tid];
      if (EVFLAG && vflag) {
        for (int r=0; r<6; r++) red_acc[r][tid]=vO[r];
        for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
          simdsync();
          if (offset < s) {
            for (int r=0; r<6; r++)
              red_acc[r][tid] += red_acc[r][tid+s];
          }
        }
        for (int r=0; r<6; r++) vO[r]=red_acc[r][tid];
      }
#else
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        fO.x += shfl_down(fO.x, s, t_per_atom);
        fO.y += shfl_down(fO.y, s, t_per_atom);
        fO.z += shfl_down(fO.z, s, t_per_atom);
        fO.w += shfl_down(fO.w, s, t_per_atom);
      }
      if (EVFLAG && vflag) {
        for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
          for (int r=0; r<6; r++)
            vO[r] += shfl_down(vO[r], s, t_per_atom);
        }
      }
#endif
    }
    if(offset == 0) {
      ansO[i] = fO;
      if (EVFLAG && vflag) {
        ansO[inum   + i].x = vO[0];
        ansO[inum   + i].y = vO[1];
        ansO[inum   + i].z = vO[2];
        ansO[inum*2 + i].x = vO[3];
        ansO[inum*2 + i].y = vO[4];
        ansO[inum*2 + i].z = vO[5];
      }
    }
  } // if ii
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}
