#include "hip/hip_runtime.h"
// **************************************************************************
//                               neighbor_gpu.cu
//                             -------------------
//                            Nitin Dhamankar (Intel)
//                              Peng Wang (Nvidia)
//                           W. Michael Brown (ORNL)
//
//  Device code for handling GPU generated neighbor lists
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : penwang@nvidia.com, brownw@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_preprocessor.h"
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#ifdef USE_OPENCL
#define tagint long
#else
#include "stdint.h"
#define tagint int64_t
#endif
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif

#ifdef NV_KERNEL
#if (__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 2)
// Issue with incorrect results in CUDA >= 11.2
#define LAL_USE_OLD_NEIGHBOR
#endif
#endif

#ifdef USE_HIP
#define LAL_USE_OLD_NEIGHBOR
#endif

__kernel void calc_cell_id(const numtyp4 *restrict x_,
                           unsigned *restrict cell_id,
                           int *restrict particle_id,
                           numtyp boxlo0, numtyp boxlo1, numtyp boxlo2,
                           numtyp i_cell_size, int ncellx, int ncelly,
                           int ncellz, int inum, int nall,
                           int cells_in_cutoff) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nall) {
    numtyp4 p;
    fetch4(p,i,pos_tex); //x_[i];

    p.x -= boxlo0;
    p.y -= boxlo1;
    p.z -= boxlo2;

    int ix = int(p.x*i_cell_size+cells_in_cutoff);
    int iy = int(p.y*i_cell_size+cells_in_cutoff);
    int iz = int(p.z*i_cell_size+cells_in_cutoff);

    int offset_lo, offset_hi;
    if (i<inum) {
      offset_lo=cells_in_cutoff;
      offset_hi=cells_in_cutoff+1;
    } else {
      offset_lo=0;
      offset_hi=1;
    }

    ix = max(ix,offset_lo);
    ix = min(ix,ncellx-offset_hi);
    iy = max(iy,offset_lo);
    iy = min(iy,ncelly-offset_hi);
    iz = max(iz,offset_lo);
    iz = min(iz,ncellz-offset_hi);

    cell_id[i] = ix+iy*ncellx+iz*ncellx*ncelly;
    particle_id[i] = i;
  }
}

__kernel void kernel_calc_cell_counts(const unsigned *restrict cell_id,
                                      int *restrict cell_counts,
                                      int nall, int ncell) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nall) {
    int id = cell_id[idx];

    // handle boundary cases
    if (idx == 0) {
      for (int i = 0; i < id + 1; i++)
        cell_counts[i] = 0;
    }
    if (idx == nall - 1) {
      for (int i = id+1; i <= ncell; i++)
        cell_counts[i] = nall;
    }

    if (idx > 0 && idx < nall) {
      int id_l = cell_id[idx-1];
      if (id != id_l) {
        for (int i = id_l+1; i <= id; i++)
          cell_counts[i] = idx;
      }
    }
  }
}

#else
#define pos_tex x_
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#define tagint long
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#endif

__kernel void transpose(__global tagint *restrict out,
                        const __global tagint *restrict in,
                        int columns_in, int rows_in)
{
  __local tagint block[BLOCK_CELL_2D][BLOCK_CELL_2D+1];

  unsigned ti=THREAD_ID_X;
  unsigned tj=THREAD_ID_Y;
  unsigned bi=BLOCK_ID_X;
  unsigned bj=BLOCK_ID_Y;

  unsigned i=bi*BLOCK_CELL_2D+ti;
  unsigned j=bj*BLOCK_CELL_2D+tj;
  if ((i<columns_in) && (j<rows_in))
    block[tj][ti]=in[j*columns_in+i];

   __syncthreads();

  i=bj*BLOCK_CELL_2D+ti;
  j=bi*BLOCK_CELL_2D+tj;
  if ((i<rows_in) && (j<columns_in))
    out[j*rows_in+i] = block[ti][tj];
}

#ifndef LAL_USE_OLD_NEIGHBOR

#define MAX_STENCIL_SIZE 25
#if !defined(MAX_SUBGROUPS_PER_BLOCK)
#define MAX_SUBGROUPS_PER_BLOCK 8
#endif

#if defined(NV_KERNEL) || defined(USE_HIP)
__device__ __constant__  int bin_stencil[MAX_STENCIL_SIZE];
#endif

__kernel void calc_neigh_list_cell(const __global numtyp4 *restrict x_,
                            const __global int *restrict cell_particle_id,
                            const __global int *restrict cell_counts,
                            __global int *nbor_list,
                            __global int *host_nbor_list,
                            __global int *host_numj,
                            int neigh_bin_size, numtyp cutoff_neigh,
                            int ncellx, int ncelly, int ncellz,
                            int inum, int nt, int nall, int t_per_atom,
                            int cells_in_cutoff,
                            const __global int *restrict cell_subgroup_counts,
                            const __global int *restrict subgroup2cell,
                            int subgroup_count,
#if defined(NV_KERNEL) || defined(USE_HIP)
                            int *not_used, __global int *error_flag)
#else
                            __constant int *bin_stencil,
                            __global int *error_flag)
#endif
{
  int tid = THREAD_ID_X;
  int bsx = BLOCK_SIZE_X;
  int simd_size = simd_size();
  int subgroup_id_local = tid / simd_size;
  int subgroup_id_global = BLOCK_ID_X * bsx / simd_size + subgroup_id_local;
  int lane_id = tid % simd_size;

#if (SHUFFLE_AVAIL == 0)
  __local int cell_list_sh[BLOCK_NBOR_BUILD];
  __local numtyp4 pos_sh[BLOCK_NBOR_BUILD];
  __local int local_cell_counts[BLOCK_NBOR_BUILD];
#endif
  __local int local_begin[(MAX_STENCIL_SIZE+1)*MAX_SUBGROUPS_PER_BLOCK];
  __local int local_counts[(MAX_STENCIL_SIZE+1)*MAX_SUBGROUPS_PER_BLOCK];

  if (subgroup_id_global < subgroup_count) {
    // identify own cell for subgroup (icell) and local atom (i) for the lane
    int icell = subgroup2cell[subgroup_id_global];
    int icell_end = cell_counts[icell+1];
    int i = cell_counts[icell] + (subgroup_id_global -
                                  cell_subgroup_counts[icell]) *
      simd_size + lane_id;

    // Get count of the number of iterations to finish all cells
    const int bin_stencil_stride = cells_in_cutoff * 2 + 1;
    const int bin_stencil_size = bin_stencil_stride * bin_stencil_stride;
    int offset = 0;
    int cell_count = 0, jcellyz, jcell_begin;
    const int offset2 = subgroup_id_local * (MAX_STENCIL_SIZE+1);
    const int niter = (bin_stencil_size - 1)/simd_size + 1;
    int end_idx = simd_size;
    for (int ni = 0; ni < niter; ni++) {
      if (ni == niter - 1)
        end_idx = bin_stencil_size - offset;
      if (lane_id < end_idx) {
        jcellyz = icell + bin_stencil[lane_id + offset];
        jcell_begin = cell_counts[jcellyz - cells_in_cutoff];
        local_begin[lane_id + offset2 + offset] = jcell_begin;
            const int local_count = cell_counts[jcellyz + cells_in_cutoff + 1] -
                                    jcell_begin;
            cell_count += local_count;
        local_counts[lane_id + offset2 + offset] = local_count;
      }
      offset += simd_size;
    }

#if (SHUFFLE_AVAIL == 0)
    local_cell_counts[tid] = cell_count;
    offset = subgroup_id_local * simd_size;
    for (unsigned int mask=simd_size/2; mask>0; mask>>=1) {
      simdsync();
      local_cell_counts[tid] += local_cell_counts[ offset + lane_id^mask ];
    }
    simdsync();
    cell_count = local_cell_counts[tid];
#else
    #pragma unroll
    for (unsigned int s=simd_size/2; s>0; s>>=1)
      cell_count += shfl_xor(cell_count, s, simd_size);
#endif

    int num_iter = cell_count;
    int remainder = num_iter % simd_size;
    if (remainder == 0) remainder = simd_size;
    if (num_iter) num_iter = (num_iter - 1) / simd_size + 1;

    numtyp4 diff;
    numtyp r2;

    int pid_i = nall, lpid_j, stride;
    numtyp4 atom_i, atom_j;
    int cnt = 0;
    __global int *neigh_counts, *neigh_list;

    if (i < icell_end)
      pid_i = cell_particle_id[i];

    if (pid_i < nt) {
      fetch4(atom_i,pid_i,pos_tex); //pos[i];
    }

    if (pid_i < inum) {
      stride=inum;
      neigh_counts=nbor_list+stride+pid_i;
      neigh_list=neigh_counts+stride+pid_i*(t_per_atom-1);
      stride=stride*t_per_atom-t_per_atom;
      nbor_list[pid_i]=pid_i;
    } else {
      stride=0;
      neigh_counts=host_numj+pid_i-inum;
      neigh_list=host_nbor_list+(pid_i-inum)*neigh_bin_size;
    }

    // loop through neighbors
    int bin_shift = 0;
    int zy = -1;
    int num_atom_cell = 0;
    int cell_pos = lane_id;
    end_idx = simd_size;
    for (int ci = 0; ci < num_iter; ci++) {
      cell_pos += simd_size;
      while (cell_pos >= num_atom_cell && zy < bin_stencil_size) {
        // Shift lane index into atom bins based on remainder from last bin
        bin_shift += num_atom_cell % simd_size;
        if (bin_shift >= simd_size) bin_shift -= simd_size;
        cell_pos = lane_id - bin_shift;
        if (cell_pos < 0) cell_pos += simd_size;
        // Move to next bin
        zy++;
        jcell_begin = local_begin[offset2 + zy];
        num_atom_cell = local_counts[offset2 + zy];
      }

      if (zy < bin_stencil_size) {
        lpid_j =  cell_particle_id[jcell_begin + cell_pos];
        fetch4(atom_j,lpid_j,pos_tex);
#if (SHUFFLE_AVAIL == 0)
        cell_list_sh[tid] = lpid_j;
        pos_sh[tid].x = atom_j.x;
        pos_sh[tid].y = atom_j.y;
        pos_sh[tid].z = atom_j.z;
      }
      simdsync();
#else
      }
#endif

      if (ci == num_iter-1) end_idx = remainder;

      for (int j = 0; j < end_idx; j++) {
#if (SHUFFLE_AVAIL == 0)
        int pid_j = cell_list_sh[offset+j]; // gather from shared memory
        diff.x = atom_i.x - pos_sh[offset+j].x;
        diff.y = atom_i.y - pos_sh[offset+j].y;
        diff.z = atom_i.z - pos_sh[offset+j].z;
#else
        int pid_j = simd_broadcast_i(lpid_j, j, simd_size);
#ifdef _DOUBLE_DOUBLE
        diff.x = atom_i.x - simd_broadcast_d(atom_j.x, j, simd_size);
        diff.y = atom_i.y - simd_broadcast_d(atom_j.y, j, simd_size);
        diff.z = atom_i.z - simd_broadcast_d(atom_j.z, j, simd_size);
#else
        diff.x = atom_i.x - simd_broadcast_f(atom_j.x, j, simd_size);
        diff.y = atom_i.y - simd_broadcast_f(atom_j.y, j, simd_size);
        diff.z = atom_i.z - simd_broadcast_f(atom_j.z, j, simd_size);
#endif
#endif

        r2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
//USE CUTOFFSQ?
        if (r2 < cutoff_neigh*cutoff_neigh && pid_j != pid_i && pid_i < nt) {
          if (cnt < neigh_bin_size) {
            cnt++;
            *neigh_list = pid_j;
            neigh_list++;
            if ((cnt & (t_per_atom-1))==0)
              neigh_list=neigh_list+stride;
          } else
            *error_flag=1;
        }
      } // for j
#if (SHUFFLE_AVAIL == 0)
      simdsync();
#endif
    } // for (ci)
    if (pid_i < nt)
      *neigh_counts = cnt;
  } // if (subgroup_id_global < subgroup_count)
}

#else

__kernel void calc_neigh_list_cell(const __global numtyp4 *restrict x_,
                                const __global int *restrict cell_particle_id,
                                const __global int *restrict cell_counts,
                                __global int *nbor_list,
                                __global int *host_nbor_list,
                                __global int *host_numj,
                                int neigh_bin_size, numtyp cell_size,
                                int ncellx, int ncelly, int ncellz,
                                int inum, int nt, int nall, int t_per_atom,
                                int cells_in_cutoff)
{
  int tid = THREAD_ID_X;
  int ix = BLOCK_ID_X + cells_in_cutoff;
  int iy = BLOCK_ID_Y % (ncelly - cells_in_cutoff*2) + cells_in_cutoff;
  int iz = BLOCK_ID_Y / (ncelly - cells_in_cutoff*2) + cells_in_cutoff;
  int bsx = BLOCK_SIZE_X;

  int icell = ix + iy*ncellx + iz*ncellx*ncelly;

  __local int cell_list_sh[BLOCK_NBOR_BUILD];
  __local numtyp4 pos_sh[BLOCK_NBOR_BUILD];

  int icell_begin = cell_counts[icell];
  int icell_end = cell_counts[icell+1];

  int nborz0 = iz-cells_in_cutoff, nborz1 = iz+cells_in_cutoff,
      nbory0 = iy-cells_in_cutoff, nbory1 = iy+cells_in_cutoff,
      nborx0 = ix-cells_in_cutoff, nborx1 = ix+cells_in_cutoff;

  numtyp4 diff;
  numtyp r2;
  int cap=ucl_ceil((numtyp)(icell_end - icell_begin)/bsx);
  for (int ii = 0; ii < cap; ii++) {
    int i = icell_begin + tid + ii*bsx;
    int pid_i = nall, pid_j, stride;
    numtyp4 atom_i, atom_j;
    int cnt = 0;
    __global int *neigh_counts, *neigh_list;

    if (i < icell_end)
      pid_i = cell_particle_id[i];

    if (pid_i < nt) {
      fetch4(atom_i,pid_i,pos_tex); //pos[i];
    }
    if (pid_i < inum) {
      stride=inum;
      neigh_counts=nbor_list+stride+pid_i;
      neigh_list=neigh_counts+stride+pid_i*(t_per_atom-1);
      stride=stride*t_per_atom-t_per_atom;
      nbor_list[pid_i]=pid_i;
    } else {
      stride=0;
      neigh_counts=host_numj+pid_i-inum;
      neigh_list=host_nbor_list+(pid_i-inum)*neigh_bin_size;
    }

    // loop through neighbors

    for (int nborz = nborz0; nborz <= nborz1; nborz++) {
      for (int nbory = nbory0; nbory <= nbory1; nbory++) {
        for (int nborx = nborx0; nborx <= nborx1; nborx++) {

          int jcell = nborx + nbory*ncellx + nborz*ncellx*ncelly;

          int jcell_begin = cell_counts[jcell];
          int jcell_end = cell_counts[jcell+1];
          int num_atom_cell = jcell_end - jcell_begin;

          // load jcell to shared memory
          int num_iter = ucl_ceil((numtyp)num_atom_cell/bsx);

          for (int k = 0; k < num_iter; k++) {
            int end_idx = min(bsx, num_atom_cell-k*bsx);

            if (tid < end_idx) {
              pid_j =  cell_particle_id[tid+k*bsx+jcell_begin];
              cell_list_sh[tid] = pid_j;
              fetch4(atom_j,pid_j,pos_tex); //[pid_j];
              pos_sh[tid].x = atom_j.x;
              pos_sh[tid].y = atom_j.y;
              pos_sh[tid].z = atom_j.z;
            }
            __syncthreads();

            if (pid_i < nt) {

              for (int j = 0; j < end_idx; j++) {
                int pid_j = cell_list_sh[j]; // gather from shared memory
                diff.x = atom_i.x - pos_sh[j].x;
                diff.y = atom_i.y - pos_sh[j].y;
                diff.z = atom_i.z - pos_sh[j].z;

                r2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
                if (r2 < cell_size*cell_size && pid_j != pid_i) {
                  cnt++;
                  if (cnt <= neigh_bin_size) {
                    *neigh_list = pid_j;
                    neigh_list++;
                    if ((cnt & (t_per_atom-1))==0)
                      neigh_list=neigh_list+stride;
                  }
                }
              }
            }
            __syncthreads();
          } // for (k)
        }
      }
    }
    if (pid_i < nt)
      *neigh_counts = cnt;
  } // for (i)
}

#endif

__kernel void kernel_special(__global int *dev_nbor,
                             __global int *host_nbor_list,
                             const __global int *host_numj,
                             const __global tagint *restrict tag,
                             const __global int *restrict nspecial,
                             const __global tagint *restrict special,
                             int inum, int nt, int max_nbors, int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=fast_mul((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid & (t_per_atom-1);

  if (ii<nt) {
    int stride;
    __global int *list, *list_end;

    int n1=nspecial[ii*3];
    int n2=nspecial[ii*3+1];
    int n3=nspecial[ii*3+2];

    int numj;
    if (ii < inum) {
      stride=inum;
      list=dev_nbor+stride+ii;
      numj=*list;
      list+=stride+fast_mul(ii,t_per_atom-1);
      stride=fast_mul(inum,t_per_atom);
      int njt=numj/t_per_atom;
      list_end=list+fast_mul(njt,stride)+(numj & (t_per_atom-1));
      list+=offset;
    } else {
      stride=1;
      list=host_nbor_list+(ii-inum)*max_nbors;
      numj=host_numj[ii-inum];
      list_end=list+fast_mul(numj,stride);
    }

    for ( ; list<list_end; list+=stride) {
      int nbor=*list;
      tagint jtag=tag[nbor];

      int offset=ii;
      for (int i=0; i<n3; i++) {
        if (special[offset]==jtag) {
          int which = 1;
          if (i>=n1)
            which++;
          if (i>=n2)
            which++;
          nbor=nbor ^ (which << SBBITS);
          *list=nbor;
        }
        offset+=nt;
      }
    }
  } // if ii
}
