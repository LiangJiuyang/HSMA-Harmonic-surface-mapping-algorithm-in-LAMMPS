#include "hip/hip_runtime.h"
// **************************************************************************
//                                   lj.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the lj/cut pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif
#else
#define pos_tex x_
#endif

__kernel void k_lj(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict lj1,
                   const __global numtyp4 *restrict lj3,
                   const int lj_types,
                   const __global numtyp *restrict sp_lj,
                   const __global int * dev_nbor,
                   const __global int * dev_packed,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (r2inv<lj1[mtype].z) {
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        force*=factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_lj_fast(const __global numtyp4 *restrict x_,
                        const __global numtyp4 *restrict lj1_in,
                        const __global numtyp4 *restrict lj3_in,
                        const __global numtyp *restrict sp_lj_in,
                        const __global int * dev_nbor,
                        const __global int * dev_packed,
                        __global acctyp4 *restrict ans,
                        __global acctyp *restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }
  __syncthreads();
  #else
  const numtyp lj1x=lj1_in[ONETYPE].x;
  const numtyp lj1y=lj1_in[ONETYPE].y;
  const numtyp cutsq=lj1_in[ONETYPE].z;
  numtyp lj3x, lj3y, lj3z;
  if (EVFLAG && eflag) {
    lj3x=lj3_in[ONETYPE].x;
    lj3y=lj3_in[ONETYPE].y;
    lj3z=lj3_in[ONETYPE].z;
  }
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    numtyp factor_lj;
    #endif

    NOUNROLL
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      numtyp cutsq=lj1[mtype].z;
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<cutsq) {
        #ifndef ONETYPE
        numtyp lj1x=lj1[mtype].x;
        numtyp lj1y=lj1[mtype].y;
        #endif

        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*r6inv*(lj1x*r6inv-lj1y);
        #ifndef ONETYPE
        force*=factor_lj;
        #endif

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          #ifndef ONETYPE
          numtyp lj3x=lj3[mtype].x;
          numtyp lj3y=lj3[mtype].y;
          numtyp lj3z=lj3[mtype].z;
          #endif
          numtyp e=r6inv*(lj3x*r6inv-lj3y);
          #ifndef ONETYPE
          energy+=factor_lj*(e-lj3z);
          #else
          energy+=(e-lj3z);
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

