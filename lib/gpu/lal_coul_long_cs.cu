#include "hip/hip_runtime.h"
// **************************************************************************
//                               coul_long_cs.cu
//                             -------------------
//                           Trung Nguyen (Northwestern)
//
//  Device code for acceleration of the coul/long/cs pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : June 2018
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( q_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( q_tex,int2);
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

// Note: EWALD_P is different from that in lal_preprocessor.h
//       acctyp is needed for these parameters
#define CS_EWALD_P (acctyp)9.95473818e-1
#define B0        (acctyp)-0.1335096380159268
#define B1        (acctyp)-2.57839507e-1
#define B2        (acctyp)-1.37203639e-1
#define B3        (acctyp)-8.88822059e-3
#define B4        (acctyp)-5.80844129e-3
#define B5        (acctyp)1.14652755e-1

#define EPSILON (acctyp)(1.0e-20)
#define EPS_EWALD (acctyp)(1.0e-6)
#define EPS_EWALD_SQR (acctyp)(1.0e-12)

__kernel void k_coul_long_cs(const __global numtyp4 *restrict x_,
                          const __global numtyp *restrict scale,
                          const int lj_types,
                          const __global numtyp *restrict sp_cl_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const numtyp cut_coulsq, const numtyp qqrd2e,
                          const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_cl[4];
  int n_stride;
  local_allocate_store_charge();

  sp_cl[0]=sp_cl_in[0];
  sp_cl[1]=sp_cl_in[1];
  sp_cl[2]=sp_cl_in[2];
  sp_cl[3]=sp_cl_in[3];

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp e_coul, virial[6];
  if (EVFLAG) {
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp qtmp; fetch(qtmp,i,q_tex);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq < cut_coulsq) {
        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;

        numtyp force,prefactor,_erfc;
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_rsqrt(r2inv);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp;
        if (factor_coul<(numtyp)1.0) {
          numtyp grij = g_ewald * (r+EPS_EWALD);
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= (r+EPS_EWALD);
          force = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
          // Additionally r2inv needs to be accordingly modified since the later
          // scaling of the overall force shall be consistent
          r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
          force *= r2inv;
        } else {
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= r;
          force = prefactor*(_erfc + EWALD_F*grij*expm2);
          force *= r2inv;
        }

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e = prefactor*_erfc;
          if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
          e_coul += e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  acctyp energy;
  if (EVFLAG) energy=(acctyp)0.0;
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}

__kernel void k_coul_long_cs_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp *restrict scale_in,
                               const __global numtyp *restrict sp_cl_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp scale[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_cl[4];
  int n_stride;
  local_allocate_store_charge();

  if (tid<4)
    sp_cl[tid]=sp_cl_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES)
    scale[tid]=scale_in[tid];

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp e_coul, virial[6];
  if (EVFLAG) {
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;

        numtyp force,prefactor,_erfc;
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_rsqrt(r2inv);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp;
        if (factor_coul<(numtyp)1.0) {
          numtyp grij = g_ewald * (r+EPS_EWALD);
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= (r+EPS_EWALD);
          force = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
          // Additionally r2inv needs to be accordingly modified since the later
          // scaling of the overall force shall be consistent
          r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
        } else {
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= r;
          force = prefactor * (_erfc + EWALD_F*grij*expm2);
        }

        force *= r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e = prefactor*_erfc;
          if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
          e_coul += e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  acctyp energy;
  if (EVFLAG) energy=(acctyp)0.0;
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}
