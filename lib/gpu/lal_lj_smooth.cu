#include "hip/hip_runtime.h"
// **************************************************************************
//                                   lj_smooth.cu
//                             -------------------
//                           Gurgen Melikyan (HSE University)
//
//  Device code for acceleration of the lj/smooth pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : gkmeliyan@edu.hse.ru
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif
#else
#define pos_tex x_
#endif

__kernel void k_lj_smooth(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict lj1,
                   const __global numtyp4 *restrict lj3,
                   const __global numtyp4 *restrict ljsw,
                   const __global numtyp2 *restrict ljsw0,
                   const int lj_types,
                   const __global numtyp *restrict sp_lj,
                   const __global int * dev_nbor,
                   const __global int * dev_packed,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp force, r6inv, factor_lj, forcelj;
    numtyp r, t, tsq, fskin;
    
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      
      int mtype=itype*lj_types+jtype;
      if (rsq<lj1[mtype].z) {
        
        numtyp r2inv=ucl_recip(rsq);
        if (rsq < lj1[mtype].w) {
          r6inv = r2inv*r2inv*r2inv;
          forcelj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        }
        else {
          r = sqrt(rsq);
          t = r - ljsw0[mtype].y;
          tsq = t*t;
          fskin = ljsw[mtype].x + ljsw[mtype].y*t +
            ljsw[mtype].z*tsq + ljsw[mtype].w*tsq*t;
          forcelj = fskin*r;
        }
        force = factor_lj*r2inv*forcelj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e;
          if (rsq < lj1[mtype].w)
            e = r6inv * (lj3[mtype].x*r6inv - lj3[mtype].y) - lj3[mtype].z;
          else
            e = ljsw0[mtype].x - ljsw[mtype].x*t -
              ljsw[mtype].y*tsq/2.0 - ljsw[mtype].z*tsq*t/3.0 -
              ljsw[mtype].w*tsq*tsq/4.0 - lj3[mtype].z;

          //numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_lj_smooth_fast(const __global numtyp4 *restrict x_,
                        const __global numtyp4 *restrict lj1_in,
                        const __global numtyp4 *restrict lj3_in,
                        const __global numtyp4 *restrict ljsw,
                        const __global numtyp2 *restrict ljsw0,
                        const __global numtyp *restrict sp_lj_in,
                        const __global int * dev_nbor,
                        const __global int * dev_packed,
                        __global acctyp4 *restrict ans,
                        __global acctyp *restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }
  __syncthreads();
  #else
  const numtyp lj1x=lj1_in[ONETYPE].x;
  const numtyp lj1y=lj1_in[ONETYPE].y;
  const numtyp cutsq=lj1_in[ONETYPE].z;
  numtyp lj3x, lj3y, lj3z;
  if (EVFLAG && eflag) {
    lj3x=lj3_in[ONETYPE].x;
    lj3y=lj3_in[ONETYPE].y;
    lj3z=lj3_in[ONETYPE].z;
  }
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp force, r6inv, factor_lj, forcelj;
    numtyp r, t, tsq, fskin;

    #endif

    NOUNROLL
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<lj1[mtype].z) {
        numtyp r2inv=ucl_recip(rsq);
        if (rsq < lj1[mtype].w) {
          r6inv = r2inv*r2inv*r2inv;
          forcelj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        }
        else {
          r = sqrt(rsq);
          t = r - ljsw0[mtype].y; //?
          tsq = t*t;
          fskin = ljsw[mtype].x + ljsw[mtype].y*t +
            ljsw[mtype].z*tsq + ljsw[mtype].w*tsq*t;
          forcelj = fskin*r;
        }
        force = factor_lj*r2inv*forcelj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e;
          if (rsq < lj1[mtype].w)
            e = r6inv * (lj3[mtype].x*r6inv - lj3[mtype].y) - lj3[mtype].z;
          else
            e = ljsw0[mtype].x - ljsw[mtype].x*t - 
              ljsw[mtype].y*tsq/2.0 - ljsw[mtype].z*tsq*t/3.0 -
              ljsw[mtype].w*tsq*tsq/4.0 - lj3[mtype].z; //???

          energy+=factor_lj*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}
