#include "hip/hip_runtime.h"
// **************************************************************************
//                               charmm.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the charmm/coul pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture(pos_tex, float4);
_texture(q_tex, float);
#else
_texture_2d(pos_tex, int4);
_texture(q_tex, int2);
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

__kernel void k_charmm(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict lj1,
                       const int lj_types,
                       const __global numtyp *restrict sp_lj,
                       const __global int *dev_nbor,
                       const __global int *dev_packed,
                       __global acctyp4 *restrict ans,
                       __global acctyp *restrict engv,
                       const int eflag, const int vflag,
                       const int inum, const int nbor_pitch,
                       const __global numtyp *restrict q_,
                       const numtyp cut_coulsq, const numtyp qqrd2e,
                       const numtyp denom_lj,
                       const numtyp denom_coul,
                       const numtyp cut_bothsq,
                       const numtyp cut_ljsq,
                       const numtyp cut_lj_innersq,
                       const numtyp cut_coul_innersq,
                       const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_bio();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cut_bothsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force_lj, force, r6inv, switch1;

        if (rsq < cut_ljsq) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
          if (rsq > cut_lj_innersq) {
            switch1 = (cut_ljsq-rsq);
            numtyp switch2 = (numtyp)12.0*rsq*switch1*(rsq-cut_lj_innersq)*
                             denom_lj;
            switch1 *= switch1;
            switch1 *= (cut_ljsq+(numtyp)2.0*rsq-(numtyp)3.0*cut_lj_innersq)*
                       denom_lj;
            switch2 *= r6inv*(lj1[mtype].z*r6inv-lj1[mtype].w);
            force_lj = force_lj*switch1+switch2;
          }
        } else
          force_lj = (numtyp)0.0;

        if (rsq < cut_coulsq) {
          numtyp rinv = ucl_rsqrt(rsq);
          fetch(forcecoul,j,q_tex);
          forcecoul *= factor_coul * qqrd2e * qtmp * rinv;
          if (rsq > cut_coul_innersq) {
            numtyp switch3 = (cut_coulsq-rsq) * (cut_coulsq-rsq) *
              (cut_coulsq + (numtyp)2.0*rsq - (numtyp)3.0*cut_coul_innersq) *
              denom_coul;
            forcecoul *= switch3;
          }
        } else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          e_coul += forcecoul;
          if (rsq < cut_ljsq) {
            numtyp e=r6inv*(lj1[mtype].z*r6inv-lj1[mtype].w);
            if (rsq > cut_lj_innersq)
              e *= switch1;
            energy+=factor_lj*e;
          }
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}

__kernel void k_charmm_fast(const __global numtyp4 *restrict x_,
                            const __global numtyp2 *restrict ljd_in,
                            const __global numtyp *restrict sp_lj_in,
                            const __global int *dev_nbor,
                            const __global int *dev_packed,
                            __global acctyp4 *restrict ans,
                            __global acctyp *restrict engv,
                            const int eflag, const int vflag,
                            const int inum, const int nbor_pitch,
                            const __global numtyp *restrict q_,
                            const numtyp cut_coulsq, const numtyp qqrd2e,
                            const numtyp denom_lj,
                            const numtyp denom_coul,
                            const numtyp cut_bothsq,
                            const numtyp cut_ljsq,
                            const numtyp cut_lj_innersq,
                            const numtyp cut_coul_innersq,
                            const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp2 ljd[MAX_BIO_SHARED_TYPES];
  __local numtyp sp_lj[8];
  int n_stride;
  local_allocate_store_bio();

  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_BIO_SHARED_TYPES)
    ljd[tid]=ljd_in[tid];
  if (tid+BLOCK_BIO_PAIR<MAX_BIO_SHARED_TYPES)
    ljd[tid+BLOCK_BIO_PAIR]=ljd_in[tid+BLOCK_BIO_PAIR];

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cut_bothsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force_lj, force, switch1;
        numtyp lj3, lj4;

        if (rsq < cut_ljsq) {
          numtyp eps = ucl_sqrt(ljd[itype].x*ljd[jtype].x);
          numtyp sig6 = (numtyp)0.5 * (ljd[itype].y+ljd[jtype].y);

          numtyp sig_r_6 = sig6*sig6*r2inv;
          sig_r_6 = sig_r_6*sig_r_6*sig_r_6;
          lj4 = (numtyp)4.0*eps*sig_r_6;
          lj3 = lj4*sig_r_6;
          force_lj = factor_lj*((numtyp)12.0 * lj3 - (numtyp)6.0 * lj4);
          if (rsq > cut_lj_innersq) {
            switch1 = (cut_ljsq-rsq);
            numtyp switch2 = (numtyp)12.0*rsq*switch1*(rsq-cut_lj_innersq)*
                             denom_lj;
            switch1 *= switch1;
            switch1 *= (cut_ljsq+(numtyp)2.0*rsq-(numtyp)3.0*cut_lj_innersq)*
                       denom_lj;
            switch2 *= lj3-lj4;
            force_lj = force_lj*switch1+switch2;
          }
        } else
          force_lj = (numtyp)0.0;

        if (rsq < cut_coulsq) {
          numtyp rinv = ucl_rsqrt(rsq);
          fetch(forcecoul,j,q_tex);
          forcecoul *= factor_coul * qqrd2e * qtmp * rinv;
          if (rsq > cut_coul_innersq) {
            numtyp switch3 = (cut_coulsq-rsq) * (cut_coulsq-rsq) *
              (cut_coulsq + (numtyp)2.0*rsq - (numtyp)3.0*cut_coul_innersq) *
              denom_coul;
            forcecoul *= switch3;
          }
        } else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          e_coul += forcecoul;
          if (rsq < cut_ljsq) {
            numtyp e=lj3-lj4;
            if (rsq > cut_lj_innersq)
              e *= switch1;
            energy+=factor_lj*e;
          }
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
}
