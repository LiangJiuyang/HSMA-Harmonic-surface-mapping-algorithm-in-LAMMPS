#include "hip/hip_runtime.h"
// **************************************************************************
//                                 vashishta.cu
//                             -------------------
//                           Anders Hafreager (UiO)
//
//  Device code for acceleration of the vashishta pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Mon June 12, 2017
//    email                : andershaf@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( param1_tex,float4);
_texture( param2_tex,float4);
_texture( param3_tex,float4);
_texture( param4_tex,float4);
_texture( param5_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture( param1_tex,int4);
_texture( param2_tex,int4);
_texture( param3_tex,int4);
_texture( param4_tex,int4);
_texture( param5_tex,int4);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define param1_tex param1
#define param2_tex param2
#define param3_tex param3
#define param4_tex param4
#define param5_tex param5
#endif

#else
#define pos_tex x_
#define param1_tex param1
#define param2_tex param2
#define param3_tex param3
#define param4_tex param4
#define param5_tex param5
#endif



#define THIRD (numtyp)0.66666666666666666667

//#define THREE_CONCURRENT

#if (SHUFFLE_AVAIL == 0)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom,       \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add3(t_per_atom, red_acc, offset, tid, f.x, f.y, f.z);      \
    if (EVFLAG && (vflag==2 || eflag==2)) {                                 \
      if (eflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_add1(t_per_atom, red_acc, offset, tid, energy);         \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_arr(6, t_per_atom, red_acc, offset, tid, virial);       \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }                                                                         \
  if (EVFLAG && (eflag || vflag)) {                                         \
    int ei=BLOCK_ID_X;                                                      \
    if (eflag!=2 && vflag!=2) {                                             \
      if (eflag) {                                                          \
        simdsync();                                                         \
        block_reduce_add1(simd_size(), red_acc, tid, energy);               \
        if (vflag) __syncthreads();                                         \
        if (tid==0) {                                                       \
          engv[ei]+=energy*(acctyp)0.5;                                     \
          ei+=ev_stride;                                                    \
        }                                                                   \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        block_reduce_arr(6, simd_size(), red_acc, tid, virial);             \
        if (tid==0) {                                                       \
          for (int r=0; r<6; r++) {                                         \
            engv[ei]+=virial[r]*(acctyp)0.5;                                \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (EVFLAG && eflag) {                                                \
        engv[ei]+=energy*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
      }                                                                     \
      if (EVFLAG && vflag) {                                                \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]+=virial[i]*(acctyp)0.5;                                  \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

#else

#if (EVFLAG == 1)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom,       \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add3(t_per_atom, f.x, f.y, f.z);                            \
    if (vflag==2 || eflag==2) {                                             \
      if (eflag)                                                            \
        simd_reduce_add1(t_per_atom,energy);                                \
      if (vflag)                                                            \
        simd_reduce_arr(6, t_per_atom,virial);                              \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }                                                                         \
  if (eflag || vflag) {                                                     \
    if (eflag!=2 && vflag!=2) {                                             \
      const int vwidth = simd_size();                                       \
      const int voffset = tid & (simd_size() - 1);                          \
      const int bnum = tid/simd_size();                                     \
      int active_subgs = BLOCK_SIZE_X/simd_size();                          \
      for ( ; active_subgs > 1; active_subgs /= vwidth) {                   \
        if (active_subgs < BLOCK_SIZE_X/simd_size()) __syncthreads();       \
        if (bnum < active_subgs) {                                          \
          if (eflag) {                                                      \
            simd_reduce_add1(vwidth, energy);                               \
            if (voffset==0) red_acc[6][bnum] = energy;                      \
          }                                                                 \
          if (vflag) {                                                      \
            simd_reduce_arr(6, vwidth, virial);                             \
            if (voffset==0)                                                 \
              for (int r=0; r<6; r++) red_acc[r][bnum]=virial[r];           \
          }                                                                 \
        }                                                                   \
                                                                            \
        __syncthreads();                                                    \
        if (tid < active_subgs) {                                           \
            if (eflag) energy = red_acc[6][tid];                            \
          if (vflag)                                                        \
            for (int r = 0; r < 6; r++) virial[r] = red_acc[r][tid];        \
        } else {                                                            \
          if (eflag) energy = (acctyp)0;                                    \
          if (vflag) for (int r = 0; r < 6; r++) virial[r] = (acctyp)0;     \
        }                                                                   \
      }                                                                     \
                                                                            \
      if (bnum == 0) {                                                      \
        int ei=BLOCK_ID_X;                                                  \
        if (eflag) {                                                        \
          simd_reduce_add1(vwidth, energy);                                 \
          if (tid==0) {                                                     \
            engv[ei]+=energy*(acctyp)0.5;                                   \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
        if (vflag) {                                                        \
          simd_reduce_arr(6, vwidth, virial);                               \
          if (tid==0) {                                                     \
            for (int r=0; r<6; r++) {                                       \
              engv[ei]+=virial[r]*(acctyp)0.5;                              \
              ei+=ev_stride;                                                \
            }                                                               \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (eflag) {                                                          \
        engv[ei]+=energy*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
      }                                                                     \
      if (vflag) {                                                          \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]+=virial[i]*(acctyp)0.5;                                  \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom,       \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1)                                                         \
    simd_reduce_add3(t_per_atom, f.x, f.y, f.z);                            \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif
#endif

__kernel void k_vashishta_short_nbor(const __global numtyp4 *restrict x_,
                                     const __global numtyp4 *restrict param4,
                                     const __global int *restrict map,
                                     const __global int *restrict elem2param,
                                     const int nelements, const int nparams,
                                     __global int * dev_nbor,
                                     const __global int * dev_packed,
                                     const int inum, const int nbor_pitch,
                                     const int t_per_atom) {
  const int ii=GLOBAL_ID_X;

  if (ii<inum) {
    const int i=dev_packed[ii];
    int nbor=ii+nbor_pitch;
    const int numj=dev_packed[nbor];
    nbor+=nbor_pitch;
    const int nbor_end=nbor+fast_mul(numj,nbor_pitch);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];
    int newj=0;

    __global int *out_list=dev_nbor+2*nbor_pitch+ii*t_per_atom;
    const int out_stride=nbor_pitch*t_per_atom-t_per_atom;

    for ( ; nbor<nbor_end; nbor+=nbor_pitch) {
      int sj=dev_packed[nbor];
      int j = sj & NEIGHMASK;
      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<param4[ijparam].x) {
        *out_list=sj;
        out_list++;
        newj++;
        if ((newj & (t_per_atom-1))==0)
          out_list+=out_stride;
      }
    } // for nbor
    dev_nbor[ii+nbor_pitch]=newj;
  } // if ii
}

__kernel void k_vashishta(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict param1,
                   const __global numtyp4 *restrict param2,
                   const __global numtyp4 *restrict param3,
                   const __global numtyp4 *restrict param4,
                   const __global numtyp4 *restrict param5,
                   const __global int *restrict map,
                   const __global int *restrict elem2param,
                   const int nelements,
                   const __global int * dev_packed,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int ev_stride) {
  const int ii=GLOBAL_ID_X;

  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    const int i=dev_packed[ii];
    int nbor=ii+nbor_pitch;
    const int numj=dev_packed[nbor];
    nbor+=nbor_pitch;
    const int nbor_end=nbor+fast_mul(numj,nbor_pitch);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor<nbor_end; nbor+=nbor_pitch) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<param4[ijparam].z) { // cutsq = param4[ijparam].z
        numtyp4 param1_ijparam; fetch4(param1_ijparam,ijparam,param1_tex);
        numtyp param1_eta=param1_ijparam.x;
        numtyp param1_lam1inv=param1_ijparam.y;
        numtyp param1_lam4inv=param1_ijparam.z;
        numtyp param1_zizj=param1_ijparam.w;

        numtyp4 param2_ijparam; fetch4(param2_ijparam,ijparam,param2_tex);
        numtyp param2_mbigd=param2_ijparam.x;
        numtyp param2_dvrc =param2_ijparam.y;
        numtyp param2_big6w=param2_ijparam.z;
        numtyp param2_heta =param2_ijparam.w;

        numtyp4 param3_ijparam; fetch4(param3_ijparam,ijparam,param3_tex);
        numtyp param3_bigh=param3_ijparam.x;
        numtyp param3_bigw=param3_ijparam.y;
        numtyp param3_dvrc=param3_ijparam.z;
        numtyp param3_c0  =param3_ijparam.w;

        numtyp r=ucl_sqrt(rsq);
        numtyp rinvsq=1.0/rsq;
        numtyp r4inv = rinvsq*rinvsq;
        numtyp r6inv = rinvsq*r4inv;

        numtyp reta = pow(r,-param1_eta);
        numtyp lam1r = r*param1_lam1inv;
        numtyp lam4r = r*param1_lam4inv;
        numtyp vc2 = param1_zizj * ucl_exp(-lam1r)/r;
        numtyp vc3 = param2_mbigd * r4inv*ucl_exp(-lam4r);

        numtyp force = (param2_dvrc*r
            - (4.0*vc3 + lam4r*vc3+param2_big6w*r6inv
               - param2_heta*reta - vc2 - lam1r*vc2)
            ) * rinvsq;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag)
          energy += (param3_bigh*reta+vc2-vc3-param3_bigw*r6inv-r*param3_dvrc+param3_c0);

        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  const int tid=THREAD_ID_X;
  store_answers_p(f,energy,virial,ii,inum,tid,1,0,eflag,vflag,ans,engv,
                  ev_stride);
}

#define threebody(delr1x, delr1y, delr1z, eflag, energy)                     \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - param_r0_ij);                               \
  numtyp gsrainv1 = param_gamma_ij * rainv1;                                 \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rinvsq2 = ucl_recip(rsq2);                                          \
  numtyp rainv2 = ucl_recip(r2 - param_r0_ik);                               \
  numtyp gsrainv2 = param_gamma_ik * rainv2;                                 \
  numtyp gsrainvsq2 = gsrainv2*rainv2/r2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - param_costheta_ijk;                                    \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = param_bigc_ijk*delcssq+1.0;                                \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = param_bigb_ijk * facexp*pcs;                               \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp frad2 = facrad*gsrainvsq2;                                          \
  numtyp facang = param_big2b_ijk * facexp*delcs/pcsinvsq;                   \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
                                                                             \
  numtyp csfac2 = rinvsq2*csfacang;                                          \
                                                                             \
  fkx = delr2x*(frad2+csfac2)-delr1x*facang12;                               \
  fky = delr2y*(frad2+csfac2)-delr1y*facang12;                               \
  fkz = delr2z*(frad2+csfac2)-delr1z*facang12;                               \
                                                                             \
  if (EVFLAG && eflag)                                                       \
    energy+=facrad;                                                          \
  if (EVFLAG && vflag) {                                                     \
    virial[0] += delr1x*fjx + delr2x*fkx;                                    \
    virial[1] += delr1y*fjy + delr2y*fky;                                    \
    virial[2] += delr1z*fjz + delr2z*fkz;                                    \
    virial[3] += delr1x*fjy + delr2x*fky;                                    \
    virial[4] += delr1x*fjz + delr2x*fkz;                                    \
    virial[5] += delr1y*fjz + delr2y*fkz;                                    \
  }                                                                          \
}

#define threebody_half(delr1x, delr1y, delr1z)                               \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - param_r0_ij);                               \
  numtyp gsrainv1 = param_gamma_ij * rainv1;                                 \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rainv2 = ucl_recip(r2 - param_r0_ik);                               \
  numtyp gsrainv2 = param_gamma_ik * rainv2;                                 \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - param_costheta_ijk;                                    \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = param_bigc_ijk*delcssq+1.0;                                \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = param_bigb_ijk * facexp*pcs;                               \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp facang = param_big2b_ijk * facexp*delcs/pcsinvsq;                   \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
}

__kernel void k_vashishta_three_center(const __global numtyp4 *restrict x_,
                                const __global numtyp4 *restrict param1,
                                const __global numtyp4 *restrict param2,
                                const __global numtyp4 *restrict param3,
                                const __global numtyp4 *restrict param4,
                                const __global numtyp4 *restrict param5,
                                const __global int *restrict map,
                                const __global int *restrict elem2param,
                                const int nelements,
                                const __global int * dev_nbor,
                                __global acctyp4 *restrict ans,
                                __global acctyp *restrict engv,
                                const int eflag, const int vflag,
                                const int inum,  const int nbor_pitch,
                                const int t_per_atom, const int evatom) {
  int n_stride;
  const int tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  local_allocate_store_three();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end;
    int offset_j=offset/t_per_atom;
    nbor_info_p(dev_nbor,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
                n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=dev_nbor[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = jx.x-ix.x;
      numtyp delr1y = jx.y-ix.y;
      numtyp delr1z = jx.z-ix.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij=param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue; // still keep this for neigh no and tpa > 1
      param_gamma_ij=param4_ijparam.y;
      param_r0_ij=param4_ijparam.w;

      int nbor_k = nbor_j-offset_j+offset_k;
      if (nbor_k<=nbor_j) nbor_k += n_stride;

      for ( ; nbor_k<nbor_end; nbor_k+=n_stride) {
        int k=dev_nbor[nbor_k];
        k &= NEIGHMASK;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[itype*nelements*nelements+ktype*nelements+ktype];
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);

        numtyp delr2x = kx.x-ix.x;
        numtyp delr2y = kx.y-ix.y;
        numtyp delr2z = kx.z-ix.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;

        param_r0sq_ik=param4_ikparam.x;
        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;

          int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;
          param_costheta_ijk=param5_ijkparam.y;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x -= fjx + fkx;
          f.y -= fjy + fky;
          f.z -= fjz + fkz;
        }
      }
    } // for nbor

    numtyp pre;
    if (evatom==1)
      pre=THIRD;
    else
      pre=(numtyp)2.0;
    energy*=pre;
    for (int i=0; i<6; i++)
      virial[i]*=pre;
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                eflag,vflag,ans,engv);
}

__kernel void k_vashishta_three_end(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict param1,
                             const __global numtyp4 *restrict param2,
                             const __global numtyp4 *restrict param3,
                             const __global numtyp4 *restrict param4,
                             const __global numtyp4 *restrict param5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_ilist,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  int n_stride;
  const int tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  local_allocate_store_three();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    int offset_j=offset/t_per_atom;
    nbor_info_p(dev_nbor,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
                n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=dev_nbor[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij = param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue; // still keep this for neigh no and tpa > 1

      param_gamma_ij=param4_ijparam.y;
      param_r0_ij = param4_ijparam.w;

      int nbor_k;
      if (gpu_nbor) nbor_k=j+nbor_pitch;
      else nbor_k=dev_ilist[j]+nbor_pitch;
      const int numk=dev_nbor[nbor_k];
      nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
      k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
      nbor_k+=offset_k;

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=dev_nbor[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; //jk

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);
        param_r0sq_ik=param4_ikparam.x;

        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; //jik
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_costheta_ijk=param5_ijkparam.y;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;

          numtyp fjx, fjy, fjz;
          //if (evatom==0) {
            threebody_half(delr1x,delr1y,delr1z);
          //} else {
          //  numtyp fkx, fky, fkz;
          //  threebody(delr1x,delr1y,delr1z,eflag,energy);
          //}

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
  } // if ii
  #ifdef THREE_CONCURRENT
  store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                eflag,vflag,ans,engv);
  #else
  store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv,NUM_BLOCKS_X);
  #endif
}

__kernel void k_vashishta_three_end_vatom(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict param1,
                             const __global numtyp4 *restrict param2,
                             const __global numtyp4 *restrict param3,
                             const __global numtyp4 *restrict param4,
                             const __global numtyp4 *restrict param5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_ilist,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  int n_stride;
  const int tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  local_allocate_store_three();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    int offset_j=offset/t_per_atom;
    nbor_info_p(dev_nbor,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
                n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=dev_nbor[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij=param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue;  // still keep this for neigh no and tpa > 1

      param_gamma_ij=param4_ijparam.y;
      param_r0_ij=param4_ijparam.w;

      int nbor_k;
      if (gpu_nbor) nbor_k=j+nbor_pitch;
      else nbor_k=dev_ilist[j]+nbor_pitch;
      const int numk=dev_nbor[nbor_k];
      nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
      k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
      nbor_k+=offset_k;

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=dev_nbor[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; // jk
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        param_r0sq_ik=param4_ikparam.x;

        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; // jik
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_costheta_ijk=param5_ijkparam.y;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    energy*=THIRD;
    for (int i=0; i<6; i++)
      virial[i]*=THIRD;
  } // if ii
  #ifdef THREE_CONCURRENT
  store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                eflag,vflag,ans,engv);
  #else
  store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv,NUM_BLOCKS_X);
  #endif
}

