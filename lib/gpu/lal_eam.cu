#include "hip/hip_runtime.h"
// **************************************************************************
//                                   eam.cu
//                             -------------------
//                   Trung Dac Nguyen, W. Michael Brown (ORNL)
//
//  Device code for acceleration of the eam pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov nguyentd@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( fp_tex,float);
_texture( rhor_sp1_tex,float4);
_texture( rhor_sp2_tex,float4);
_texture( frho_sp1_tex,float4);
_texture( frho_sp2_tex,float4);
_texture( z2r_sp1_tex,float4);
_texture( z2r_sp2_tex,float4);
#else
_texture( pos_tex,int4);
_texture( fp_tex,int2);
_texture( rhor_sp1_tex,int4);
_texture( rhor_sp2_tex,int4);
_texture( frho_sp1_tex,int4);
_texture( frho_sp2_tex,int4);
_texture( z2r_sp1_tex,int4);
_texture( z2r_sp2_tex,int4);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define fp_tex fp_
#define rhor_sp1_tex rhor_spline1
#define rhor_sp2_tex rhor_spline2
#define frho_sp1_tex frho_spline1
#define frho_sp2_tex frho_spline2
#define z2r_sp1_tex z2r_spline1
#define z2r_sp2_tex z2r_spline2
#endif

#else

#define pos_tex x_
#define fp_tex fp_
#define rhor_sp1_tex rhor_spline1
#define rhor_sp2_tex rhor_spline2
#define frho_sp1_tex frho_spline1
#define frho_sp2_tex frho_spline2
#define z2r_sp1_tex z2r_spline1
#define z2r_sp2_tex z2r_spline2

#endif

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MAX(A,B) ((A) > (B) ? (A) : (B))

#if (SHUFFLE_AVAIL == 0)

#define local_allocate_store_energy_fp()                                    \
    __local acctyp red_acc[BLOCK_PAIR];

#define store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,           \
                        eflag,vflag,engv,rdrho,nrho,i,rhomax,tfrho)         \
  if (t_per_atom>1) {                                                       \
    red_acc[tid]=rho;                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s)                                                       \
         red_acc[tid] += red_acc[tid+s];                                    \
      }                                                                     \
      rho=red_acc[tid];                                                     \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    numtyp p = rho*rdrho + (numtyp)1.0;                                     \
    int m=p;                                                                \
    m = MAX(1,MIN(m,nrho-1));                                               \
    p -= m;                                                                 \
    p = MIN(p,(numtyp)1.0);                                                 \
    int index = tfrho*(nrho+1)+m;                                           \
    numtyp4 coeff; fetch4(coeff,index,frho_sp1_tex);                        \
    numtyp fp = (coeff.x*p + coeff.y)*p + coeff.z;                          \
    fp_[i]=fp;                                                              \
    if (EVFLAG && eflag) {                                                  \
      fetch4(coeff,index,frho_sp2_tex);                                     \
      energy = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;             \
      if (rho > rhomax) energy += fp*(rho-rhomax);                          \
      engv[ii]=energy;                                                      \
    }                                                                       \
  }

#define local_allocate_store_answers_eam()                                  \
    __local acctyp red_acc[6][BLOCK_PAIR];

#define store_answers_eam(f, energy, virial, ii, inum, tid, t_per_atom,     \
                      offset, elag, vflag, ans, engv)                       \
  if (t_per_atom>1) {                                                       \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (EVFLAG && vflag) {                                                  \
      simdsync();                                                           \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        simdsync();                                                         \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    int ei=ii;                                                              \
    if (EVFLAG && eflag) {                                                  \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (EVFLAG && vflag) {                                                  \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#else

#define local_allocate_store_energy_fp()

#define store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,           \
                        eflag,vflag,engv,rdrho,nrho,i,rhomax,tfrho)         \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1)                           \
      rho += shfl_down(rho, s, t_per_atom);                                 \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    numtyp p = rho*rdrho + (numtyp)1.0;                                     \
    int m=p;                                                                \
    m = MAX(1,MIN(m,nrho-1));                                               \
    p -= m;                                                                 \
    p = MIN(p,(numtyp)1.0);                                                 \
    int index = tfrho*(nrho+1)+m;                                           \
    numtyp4 coeff; fetch4(coeff,index,frho_sp1_tex);                        \
    numtyp fp = (coeff.x*p + coeff.y)*p + coeff.z;                          \
    fp_[i]=fp;                                                              \
    if (EVFLAG && eflag) {                                                  \
      fetch4(coeff,index,frho_sp2_tex);                                     \
      energy = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;             \
      if (rho > rhomax) energy += fp*(rho-rhomax);                          \
      engv[ii]=energy;                                                      \
    }                                                                       \
  }

#define local_allocate_store_answers_eam()

#define store_answers_eam(f, energy, virial, ii, inum, tid, t_per_atom,     \
                          offset, eflag, vflag, ans, engv)                  \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      f.x += shfl_down(f.x, s, t_per_atom);                                 \
      f.y += shfl_down(f.y, s, t_per_atom);                                 \
      f.z += shfl_down(f.z, s, t_per_atom);                                 \
      if (EVFLAG) energy += shfl_down(energy, s, t_per_atom);               \
    }                                                                       \
    if (EVFLAG && vflag) {                                                  \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        for (int r=0; r<6; r++)                                             \
          virial[r] += shfl_down(virial[r], s, t_per_atom);                 \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    int ei=ii;                                                              \
    if (EVFLAG && eflag) {                                                  \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (EVFLAG && vflag) {                                                  \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#endif

__kernel void k_energy(const __global numtyp4 *restrict x_,
                       const __global int2 *restrict type2rhor_z2r,
                       const __global int *restrict type2frho,
                       const __global numtyp4 *restrict rhor_spline2,
                       const __global numtyp4 *restrict frho_spline1,
                       const __global numtyp4 *restrict frho_spline2,
                       const __global int *dev_nbor,
                       const __global int *dev_packed,
                       __global numtyp *restrict fp_,
                       __global acctyp *restrict engv,
                       const int eflag, const int inum, const int nbor_pitch,
                       const int ntypes,  const numtyp cutforcesq,
                       const numtyp rdr, const numtyp rdrho,
                       const numtyp rhomax, const int nrho,
                       const int nr, const int t_per_atom) {
  int tid, ii, offset, i, itype;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_energy_fp();

  acctyp rho = (acctyp)0;
  acctyp energy;
  if (EVFLAG && eflag) energy=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        int mtype = jtype*ntypes+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        numtyp4 coeff; fetch4(coeff,index,rhor_sp2_tex);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor
  } // if ii
  const numtyp tfrho=type2frho[itype];
  store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
                  eflag,vflag,engv,rdrho,nrho,i,rhomax,tfrho);
}

__kernel void k_energy_fast(const __global numtyp4 *restrict x_,
                            const __global int2 *restrict type2rhor_z2r_in,
                            const __global int *restrict type2frho_in,
                            const __global numtyp4 *restrict rhor_spline2,
                            const __global numtyp4 *restrict frho_spline1,
                            const __global numtyp4 *restrict frho_spline2,
                            const __global int *dev_nbor,
                            const __global int *dev_packed,
                            __global numtyp *restrict fp_,
                            __global acctyp *restrict engv,
                            const int eflag,  const int inum,
                            const int nbor_pitch, const int ntypes,
                            const numtyp cutforcesq,  const numtyp rdr,
                            const numtyp rdrho, const numtyp rhomax,
                            const int nrho, const int nr,
                            const int t_per_atom) {
  int tid, ii, offset, i, itype;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local int type2frho[MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }
  if (tid<MAX_SHARED_TYPES) {
    type2frho[tid]=type2frho_in[tid];
  }
  __syncthreads();
  #else
  const numtyp type2rhor_z2rx=
    type2rhor_z2r_in[ONETYPE*MAX_SHARED_TYPES+ONETYPE].x;
  const numtyp tfrho=type2frho_in[ONETYPE];
  #endif

  int n_stride;
  local_allocate_store_energy_fp();

  acctyp rho = (acctyp)0;
  acctyp energy;
  if (EVFLAG && eflag) energy=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    itype=ix.w;
    #endif

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        #ifndef ONETYPE
        int jtype=fast_mul((int)MAX_SHARED_TYPES,jx.w);
        int mtype = jtype+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        #else
        int index = type2rhor_z2rx*(nr+1)+m;
        #endif
        numtyp4 coeff; fetch4(coeff,index,rhor_sp2_tex);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor
  } // if ii
  #ifndef ONETYPE
  const numtyp tfrho=type2frho[itype];
  #endif
  store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
                  eflag,vflag,engv,rdrho,nrho,i,rhomax,tfrho);
}

__kernel void k_eam(const __global numtyp4 *restrict x_,
                    const __global numtyp *fp_,
                    const __global int2 *type2rhor_z2r,
                    const __global numtyp4 *rhor_spline1,
                    const __global numtyp4 *z2r_spline1,
                    const __global numtyp4 *z2r_spline2,
                    const __global int *dev_nbor,
                    const __global int *dev_packed,
                    __global acctyp4 *ans,
                    __global acctyp *engv,
                    const int eflag, const int vflag,  const int inum,
                    const int nbor_pitch, const int ntypes,
                    const numtyp cutforcesq,  const numtyp rdr, const int nr,
                    const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_answers_eam();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp ifp; fetch(ifp,i,fp_tex);  //fp_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        int mtype,index;
        numtyp4 coeff;

        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = jtype*ntypes+itype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        fetch4(coeff,index,z2r_sp1_tex);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        fetch4(coeff,index,z2r_sp2_tex);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;

        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip;
        fetch(psip,j,fp_tex);
        psip = ifp*rhojp + psip*rhoip + phip;
        numtyp force = -psip*recip;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          energy += phi;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                    ans,engv);
}

__kernel void k_eam_fast(const __global numtyp4 *x_,
                         const __global numtyp *fp_,
                         const __global int2 *type2rhor_z2r_in,
                         const __global numtyp4 *rhor_spline1,
                         const __global numtyp4 *z2r_spline1,
                         const __global numtyp4 *z2r_spline2,
                         const __global int *dev_nbor,
                         const __global int *dev_packed,
                         __global acctyp4 *ans,
                         __global acctyp *engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch, const numtyp cutforcesq,
                         const numtyp rdr, const int nr, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }
  __syncthreads();
  #else
  const int oi=ONETYPE*MAX_SHARED_TYPES+ONETYPE;
  const numtyp type2rhor_z2rx=type2rhor_z2r_in[oi].x;
  const numtyp type2rhor_z2ry=type2rhor_z2r_in[oi].y;
  #endif

  int n_stride;
  local_allocate_store_answers_eam();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp ifp; fetch(ifp,i,fp_tex); //fp_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int jw=jx.w;
      int jtype=fast_mul((int)MAX_SHARED_TYPES,jw);
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        numtyp4 coeff;
        #ifndef ONETYPE
        int mtype;
        #endif
        int index;

        #ifndef ONETYPE
        mtype = itype+jw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        #else
        index = type2rhor_z2rx*(nr+1)+m;
        #endif
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;

        #ifndef ONETYPE
        mtype = jtype+iw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        #else
        index = type2rhor_z2rx*(nr+1)+m;
        #endif
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;

        #ifndef ONETYPE
        mtype = itype+jw;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        #else
        index = type2rhor_z2ry*(nr+1)+m;
        #endif
        fetch4(coeff,index,z2r_sp1_tex);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        fetch4(coeff,index,z2r_sp2_tex);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;

        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip;
        fetch(psip,j,fp_tex);
        psip = ifp*rhojp + psip*rhoip + phip;
        numtyp force = -psip*recip;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          energy += phi;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                    ans,engv);
}

